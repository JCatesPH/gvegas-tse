#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#include "cutil_inline.h"

#include "vegas.h" 
#include "vegasconst.h"
#include "kernels.h"

#include "gvegas.h"

double getrusage_sec();

void gVegas(double& avgi, double& sd, double& chi2a)
{

   for (int j=0;j<ndim;j++) {
      xi[j][0] = 1.f;
   }

   // entry vegas1

   it = 0;

   // entry vegas2
   nd = nd_max;
   ng = 1;
   
   npg = 0;
   std::cout<<"mds = "<<mds<<std::endl;
   if (mds!=0) {
      
      std::cout<<"ncall, ndim = "<<ncall<<", "<<ndim<<std::endl;
      ng = (int)pow((0.5*(double)ncall),1./(double)ndim);
      mds = 1;
      //      printf("ng = %d\n",ng);
      if (2*ng>=nd_max) {
         mds = -1;
         npg = ng/nd_max+1;
         nd = ng/npg;
         ng = npg*nd;
      }
      
   }
   std::cout<<"ng = "<<ng<<std::endl;
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &ndim, sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_ng),   &ng,   sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_nd),   &nd,   sizeof(int)));
   hipDeviceSynchronize(); // wait for synchronize

   nCubes = (unsigned)(pow(ng,ndim));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_nCubes), &nCubes, sizeof(nCubes)));
   hipDeviceSynchronize(); // wait for synchronize

   npg = ncall/nCubes;
   if (npg<2) npg = 2;
   calls = (double)(npg*nCubes);

   unsigned nCubeNpg = nCubes*npg;
   
   //   std::cout<<"nCubes= "<<nCubes<<std::endl;
   //   std::cout<<"nCubeNpg= "<<nCubeNpg<<std::endl;

   if (nprn!=0) {
      // tsi = sqrt(tsi);
      std::cout<<std::endl;
      std::cout<<" << vegas internal parameters >>"<<std::endl;
      std::cout<<"            ng: "<<std::setw(5)<<ng<<std::endl;
      std::cout<<"            nd: "<<std::setw(5)<<nd<<std::endl;
      std::cout<<"           npg: "<<std::setw(5)<<npg<<std::endl;
      std::cout<<"        nCubes: "<<std::setw(12)<<nCubes<<std::endl;
      std::cout<<"    nCubes*npg: "<<std::setw(12)<<nCubeNpg<<std::endl;
   }
   
   dxg = 1.f/(float)ng;
   double dnpg = (double)npg;
   double dv2g = calls*calls*pow(dxg,ndim)*pow(dxg,ndim)/(dnpg*dnpg*(dnpg-1.));
   xnd = (float)nd;
   dxg *= xnd;
   xjac = 1.f/(float)calls;
   for (int j=0;j<ndim;j++) {
      dx[j] = xu[j]-xl[j];
      xjac *= dx[j];
   }

   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_npg),  &npg,  sizeof(int)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xjac), &xjac, sizeof(float)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_dxg),  &dxg,  sizeof(float)));
   hipDeviceSynchronize(); // wait for synchronize

   ndo = 1;

   if (nd!=ndo) {

      double rc = (double)ndo/xnd;
      
      for (int j=0;j<ndim;j++) {
         
         int k = -1;
         double xn = 0.;
         double dr = 0.;
         int i = k;
         k++;
         dr += 1.;
         double xo = xn;
         xn = xi[j][k];
         //         printf("xn = %g\n",xn);
         while (i<nd-1) {

            while (dr<=rc) {
               k++;
               dr += 1.;
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn - (xn-xo)*dr;
         }
         
         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      ndo = nd;
      
   }

   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xl), xl, sizeof(xl)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_dx), dx, sizeof(dx)));
   cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
   hipDeviceSynchronize(); // wait for synchronize

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << input parameters for vegas >>"<<std::endl;
      std::cout<<"     ndim ="<<std::setw(3)<<ndim
               <<"   ncall ="<<std::setw(10)<<(int)calls<<std::endl;
      std::cout<<"     it   =  0"
               <<"   itmx ="<<std::setw(5)<<itmx<<std::endl;
      std::cout<<"     acc  = "<<std::fixed
               <<std::setw(9)<<std::setprecision(3)<<acc<<std::endl;
      std::cout<<"     mds  ="<<std::setw(3)<<mds
               <<"   nd = "<<std::setw(4)<<nd<<std::endl;
      for (int j=0;j<ndim;j++) {
         std::cout<<"    (xl,xu)= ( "<<std::setw(6)<<std::fixed
                  <<xl[j]<<" , "<<xu[j]<<" )"<<std::endl;
      }
      
   }

   // entry vegas3

   it = 0;
   si = 0.;
   si2 = 0.;
   swgt = 0.;
   schi = 0.;
   //   int iflag;
   // main integration loop

   //   std::cout<<"nBlockSize = "<<nBlockSize<<std::endl;
   //--------------------------
   //  Set up kernel vaiables
   //--------------------------
   const int nGridSizeMax =  65535;
   
   dim3 ThBk(nBlockSize);

   int nGridSizeX, nGridSizeY;
   int nBlockTot = (nCubeNpg-1)/nBlockSize+1;
//   std::cout<<"nBlockTot = "<<nBlockTot<<std::endl;
   nGridSizeY = (nBlockTot-1)/nGridSizeMax+1;
   nGridSizeX = (nBlockTot-1)/nGridSizeY+1;
//   std::cout<<"nGridSize (x,y) = "<<nGridSizeX<<", "<<nGridSizeY<<std::endl;
   dim3 BkGd(nGridSizeX, nGridSizeY);

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << kernel parameters for CUDA >>"<<std::endl;
      std::cout<<"       Block size           ="<<std::setw(7)<<ThBk.x<<std::endl;
      std::cout<<"       Grid size            ="<<std::setw(7)<<BkGd.x
               <<" x "<<BkGd.y<<std::endl;
      int nThreadsTot = ThBk.x*BkGd.x*BkGd.y;
      std::cout<<"     Actual Number of calls ="<<std::setw(12)
               <<nThreadsTot<<std::endl;
      std::cout<<"   Required Number of calls ="<<std::setw(12)
               <<nCubeNpg<<" ( "<<std::setw(6)<<std::setprecision(2)
               <<100.*(double)nCubeNpg/(double)nThreadsTot<<"%)"<<std::endl;
      std::cout<<std::endl;
   }
      
   // allocate Fval
   int sizeFval = nCubeNpg*sizeof(float);
//   std::cout<<"sizeFval = "<<sizeFval<<std::endl;

   // CPU
   float* hFval;
   cutilSafeCall(hipHostMalloc((void**)&hFval, sizeFval));
   memset(hFval, '\0', sizeFval);

   // GPU
   float* gFval;
   cutilSafeCall(hipMalloc((void**)&gFval, sizeFval));

   // allocate IAval
   //   int sizeIAval = nCubeNpg*ndim*sizeof(unsigned short);
   int sizeIAval = nCubeNpg*ndim*sizeof(int);
//   std::cout<<"sizeIAval = "<<sizeIAval<<std::endl;

   // CPU
   //unsigned short* hIAval;
   int* hIAval;
   cutilSafeCall(hipHostMalloc((void**)&hIAval, sizeIAval));
   //unsigned short* hIAval =
   //  (unsigned short*)calloc(nCubeNpg*ndim, sizeof(unsigned short));
   memset(hIAval, '\0', sizeIAval);

   // GPU
   // unsigned short* gIAval;
   int* gIAval;
   cutilSafeCall(hipMalloc((void**)&gIAval, sizeIAval));

   double startVegasCall, endVegasCall;
   double startVegasMove, endVegasMove;
   double startVegasFill, endVegasFill;
   double startVegasRefine, endVegasRefine;

   do {
      
      it++;

//      std::cout<<"call gVegasCallFunc: it = "<<it<<std::endl;
      startVegasCall = getrusage_sec();
      gVegasCallFunc<<<BkGd, ThBk>>>(gFval, gIAval);
      hipDeviceSynchronize(); // wait for synchronize
      endVegasCall = getrusage_sec();
      timeVegasCall += endVegasCall-startVegasCall;

      startVegasMove = getrusage_sec();
      cutilSafeCall(hipMemcpy(hFval, gFval,  sizeFval,
                               hipMemcpyDeviceToHost));

      cutilSafeCall(hipMemcpy(hIAval, gIAval,  sizeIAval,
                               hipMemcpyDeviceToHost));
      endVegasMove = getrusage_sec();
      timeVegasMove += endVegasMove-startVegasMove;

// *****************         

      startVegasFill = getrusage_sec();

      ti = 0.;
      tsi = 0.;

      double d[ndim_max][nd_max];

      for (int j=0;j<ndim;++j) {
         for (int i=0;i<nd;++i) {
            d[j][i] = 0.;
         }
      }

      for (unsigned ig=0;ig<nCubes;ig++) {
         double fb = 0.;
         double f2b = 0.;
         for (int ipg=0;ipg<npg;ipg++) {
            int idx = npg*ig+ipg;
            double f = (double)hFval[idx];
//            std::cout<<"idx,f = "<<idx<<", "<<std::scientific
//                     <<std::setw(10)<<std::setprecision(5)<<f<<std::endl;
            double f2 = f*f;
            fb += f;
            f2b += f2;
            /*
            for (int idim=0;idim<ndim;idim++) {
               int iaj = hIAval[idim*nCubeNpg+idx];
               d[idim][iaj] += f2;
            }
            */
         }
         f2b = sqrt(f2b*npg);
         f2b = (f2b-fb)*(f2b+fb);
         ti += fb;
         tsi += f2b;
         if (mds<0) {
            for (int idim=0;idim<ndim;idim++) {
               int idx = npg*ig;
               int iaj = hIAval[idim*nCubeNpg+idx];
               d[idim][iaj] += f2b;
            }
         }
      }

//      std::cout<<"mds = "<<mds<<std::endl;
      if (mds>0) {
         //         std::cout<<"ndim = "<<ndim<<std::endl;
         for (int idim=0;idim<ndim;idim++) {
            //            std::cout<<"idim = "<<idim<<std::endl;
            for (int idx=0;idx<nCubeNpg;idx++) {
               //               std::cout<<"idx = "<<idx<<std::endl;
               int iaj = hIAval[idim*nCubeNpg+idx];
               //               std::cout<<"iaj = "<<iaj<<std::endl;
               double f = (double)hFval[idx];
               //               std::cout<<"f = "<<f<<std::endl;
               double f2 = f*f;
               d[idim][iaj] += f2;
               //               std::cout<<"idim, iaj, idx, f = "<<idim<<", "<<iaj
               //                        <<", "<<idx<<", "<<f<<std::endl;
            }
         }
      }

      endVegasFill = getrusage_sec();
      timeVegasFill += endVegasFill-startVegasFill;

      tsi *= dv2g;
      double ti2 = ti*ti;
      double wgt = ti2/tsi;
      si += ti*wgt;
      si2 += ti2;
      swgt += wgt;
      schi += ti2*wgt;
      avgi = si/swgt;
      sd = swgt*it/si2;
      chi2a = 0.;
      if (it>1) chi2a = sd*(schi/swgt-avgi*avgi)/((double)it-1.);
      sd = sqrt(1./sd);
      
      if (nprn!=0) {
         tsi = sqrt(tsi);
         std::cout<<std::endl;
         std::cout<<" << integration by vegas >>"<<std::endl;
         std::cout<<"     iteration no. "<<std::setw(4)<<it
                  <<"   integral=  "<<ti<<std::endl;
         std::cout<<"                          std dev  = "<<tsi<<std::endl;
         std::cout<<"     accumulated results: integral = "<<avgi<<std::endl;
         std::cout<<"                          std dev  = "<<sd<<std::endl;
	 if (it > 1) {
            std::cout<<"                          chi**2 per it'n = "
                     <<std::setw(10)<<std::setprecision(4)<<chi2a<<std::endl;
         }
         if (nprn<0) {
            for (int j=0;j<ndim;j++) {
               std::cout<<"   == data for axis "
                        <<std::setw(2)<<j<<" --"<<std::endl;
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce"<<std::endl;
               /*
               for (int i=0;i<nd;i+=3) {
                  std::cout<<std::setw(6)<<std::setprecision(2)<<std::setfill(' ')
                           <<xi[j][i]<<" "<<di[j][i]<<" "<<d[j][i];
                  std::cout<<std::setw(6)<<std::setprecision(2)
                           <<xi[j][i+1]<<" "<<di[j][i+1]<<" "<<d[j][i+1];
                  std::cout<<std::setw(6)<<std::setprecision(2)
                           <<xi[j][i+2]<<" "<<di[j][i+2]<<" "<<d[j][i+2]
                           <<std::endl;
                           }
               */
            }
         }
      }

      // refine grid

      startVegasRefine = getrusage_sec();

      /*
      for (int ii=0;ii<ndim;ii++) {
         for (int jj=0;jj<nd;jj++) {
            std::cout<<"d["<<ii<<"]["<<jj<<"] = "<<std::scientific
                     <<d[ii][jj]<<std::endl;
         }
      }
      */
      
      double r[nd_max];
      double dt[ndim_max];
      for (int j=0;j<ndim;j++) {
         double xo = d[j][0];
         double xn = d[j][1];
         d[j][0] = 0.5*(xo+xn);
         dt[j] = d[j][0];
         for (int i=1;i<nd-1;i++) {
            d[j][i] = xo+xn;
            xo = xn;
            xn = d[j][i+1];
            d[j][i] = (d[j][i]+xn)/3.;
            dt[j] += d[j][i];
         }
         d[j][nd-1] = 0.5*(xn+xo);
         dt[j] += d[j][nd-1];
      }
      
      for (int j=0;j<ndim;j++) {
         double rc = 0.;
         for (int i=0;i<nd;i++) {
            r[i] = 0.;
            if (d[j][i]>0.) {
               double xo = dt[j]/d[j][i];
               if (!isinf(xo)) 
                  r[i] = pow(((xo-1.)/xo/log(xo)),alph);
            }
            rc += r[i];
         }
         rc /= xnd;
         int k = -1;
         double xn = 0.;
         double dr = xn;
         int i = k;
         k++;
         dr += r[k];
         double xo = xn;
         xn = xi[j][k];
         
         do {

            while (dr<=rc) {
               k++;
               dr += r[k];
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn-(xn-xo)*dr/r[k];

         } while (i<nd-2);

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
      hipDeviceSynchronize(); // wait for synchronize

      endVegasRefine = getrusage_sec();
      timeVegasRefine += endVegasRefine-startVegasRefine;
      
//      std::cout<<"The end of main loop: it, sd/avgi = "<<it<<", "
//               <<sd/fabs(avgi)<<std::endl;
      
   } while (it<itmx && acc*fabs(avgi)<sd);


   cutilSafeCall(hipHostFree(hFval));
   cutilSafeCall(hipFree(gFval));

   cutilSafeCall(hipHostFree(hIAval));
//   free(hIAval);
   cutilSafeCall(hipFree(gIAval));

   //   std::cout<<"ng = "<<ng<<std::endl;
}
