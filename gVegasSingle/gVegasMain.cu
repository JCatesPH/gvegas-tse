#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
// #include <cutil_inline.h>
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

double getrusage_sec()
{
   struct rusage t;
   struct timeval tv;
   getrusage(RUSAGE_SELF, &t);
   tv = t.ru_utime;
   return tv.tv_sec + (double)tv.tv_usec*1e-6;
}

int main(int argc, char** argv)
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n="ncall0" -i="itmx0" -a="nacc" -b="nBlockSize0"
   //
   // parameters:
   //   ncall = 1024*ncall0
   //   itmx  = itmx0
   //   acc   = nacc*0.00001f
   //   nBlockSize = nBlockSize0
   //

   int ncall0 = 256;
   int itmx0 = 10;
   int nacc  = 1;
   int nBlockSize0 = 256;

   // cutGetCmdLineArgumenti(argc, (const char**)argv, "n", &ncall0);
   // cutGetCmdLineArgumenti(argc, (const char**)argv, "i", &itmx0);
   // cutGetCmdLineArgumenti(argc, (const char**)argv, "a", &nacc);
   // cutGetCmdLineArgumenti(argc, (const char**)argv, "b", &nBlockSize0);

   ncall = ncall0*1024;
   itmx = itmx0;
   acc = (float)nacc*0.00001f;
   nBlockSize = nBlockSize0;

   hipSetDevice(0);

   mds = 1;
   ndim = 8;
   
   ng = 0;
   npg = 0;

   for (int i=0;i<ndim;i++) {
      xl[i] = 0.;
      xu[i] = 1.;
   }
   
   nprn = 1;
//   nprn = -1;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   gVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   hipDeviceReset();

   return 0;
}
