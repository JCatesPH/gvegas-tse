#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   float result = 0.f;
   float T = 1.f;

   /*
   for (int i=0;i<7;i++) {
      value += rx[i];
   }
   */

   value = rx[0] + rx[1] + rx[2] + rx[3] + rx[4] + rx[5] + rx[6];
   // result = cosf(value);

   result = sqrtf(2 / 3.14159) * value * value * expf(-value * value / (2 * T * T)) / (T * T * T);

   return result;

}

/*
{

   float value = 1.f;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.f*rx[i];
   }
   return value;

}
*/