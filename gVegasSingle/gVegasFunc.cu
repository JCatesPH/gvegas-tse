#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   for (int i=0;i<7;i++) {
      value += rx[i];
   }
   return __cosf(value);

}

/*
{

   float value = 1.f;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.f*rx[i];
   }
   return value;

}
*/