#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   float cosval = 0.f;

   for (int i=0;i<7;i++) {
      value += rx[i];
   }
   
   cosval = __cosf(value);

   return cosval;
   
}

/*
{

   float value = 1.f;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.f*rx[i];
   }
   return value;

}
*/