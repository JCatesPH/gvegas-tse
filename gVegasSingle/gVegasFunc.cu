#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   float cosval = 0.f;

   /*
   for (int i=0;i<7;i++) {
      value += rx[i];
   }
   */

   value = rx[0] + rx[1] + rx[2] + rx[3] + rx[4] + rx[5] + rx[6];
   cosval = cosf(value);

   return cosval;

}

/*
{

   float value = 1.f;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.f*rx[i];
   }
   return value;

}
*/