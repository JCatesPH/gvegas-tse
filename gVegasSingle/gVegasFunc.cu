#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   for (int i=0;i<g_ndim;i++) {
      value += rx[i];
   }
   return cosf(value);

}

/*
{

   float value = 1.f;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.f*rx[i];
   }
   return value;

}
*/