#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

__device__
float heaviside(float x, float z)
{
    if (x < z)
    {
        return 0.f;
    }
    else
    {
        return 1.f;
    }
    
}

/*
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(5*sizeof(hipFloatComplex));

    int i = 0;
    for (int j=0; j<10; j=j+2) {
        vector[i] = make_hipFloatComplex(rx[j], rx[j+1]);
        i++;
    }

    for (int j=0; j<5; j++) {
        sum = hipCaddf(sum, vector[j]);
    }
    
    free(vector);

    return hipCrealf(sum);
}
*/

__device__
float func(float* rx, float wgt)
{
    hipFloatComplex z1 = make_hipFloatComplex(rx[0], rx[1]);
    hipFloatComplex z2 = make_hipFloatComplex(rx[2], rx[3]);
    
    hipFloatComplex result;

    result = hipCdivf(z1, z2);

    return hipCrealf(result);
}