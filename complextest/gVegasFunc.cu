#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

__device__
float heaviside(float x, float z)
{
    if (x < z)
    {
        return 0.f;
    }
    else
    {
        return 1.f;
    }
    
}

// ztest10: Bessel function test
__device__
float func(float* rx, float wgt)
{
    return jnf(0, rx[0]+rx[1]);
}


/*

// ztest8-9: complex vectors using for-loop assignment and operations.
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(4*sizeof(hipFloatComplex));

    //float fermi = heaviside(rx[0], 5);

    //vector[0] = make_hipFloatComplex(rx[0], rx[1]);
    //vector[1] = make_hipFloatComplex(rx[2], rx[3]);
    //vector[2] = make_hipFloatComplex(rx[4], rx[5]);
    //vector[3] = make_hipFloatComplex(rx[6], rx[7]);

    for(int i=0; i<4; i++)
    {
        vector[i] = make_hipFloatComplex(rx[2*i], rx[2*i+1]);
    }
    
    for(int i=0; i<4; i=i+2)
    {
        sum = hipCaddf(sum, hipCdivf(vector[i], vector[i+1]));
    }

    //sum = hipCaddf(vector[0], vector[1]);
    
    free(vector);

    return hipCrealf(sum);
}

*/

/*

// ztest7: Testing other device function calls.
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(2*sizeof(hipFloatComplex));

    float fermi = heaviside(rx[0], 5);

    vector[0] = make_hipFloatComplex(fermi, rx[1]);
    vector[1] = make_hipFloatComplex(rx[2], rx[3]);
    
    sum = hipCaddf(vector[0], vector[1]);
    
    free(vector);

    return hipCrealf(sum);
}

*/

/*

// ztest6: Testing complex vectors.
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(2*sizeof(hipFloatComplex));

    vector[0] = make_hipFloatComplex(rx[0], rx[1]);
    vector[1] = make_hipFloatComplex(rx[2], rx[3]);
    
    sum = hipCaddf(vector[0], vector[1]);
    
    free(vector);

    return hipCrealf(sum);
}

*/

/*

// ztest5, dbltest
__device__
float func(float* rx, float wgt)
{
    double sum = 0;
    double *vector; 
    vector = (double*)malloc(4*sizeof(double));

    for (int j=0; j<4; j++) {
        vector[j] = rx[j];
    }
    
    for (int j=0; j<4; j++) {
        sum += vector[j];
    }

    free(vector);

    return (float)sum;
}
*/

/*

// ztest2/3/4: Testing complex math operations.
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex z1 = make_hipFloatComplex(rx[0], rx[1]);
    hipFloatComplex z2 = make_hipFloatComplex(rx[2], rx[3]);
    
    hipFloatComplex result;

    result = hipCdivf(z1, z2);

    return A * hipCrealf(result);
}

*/


/*

// ztest1
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(1*sizeof(hipFloatComplex));

    // int i = 0;
    for (int j=0; j<3; j++) {
        vector[j] = make_hipFloatComplex(rx[0], rx[1]);
    // i++;
    }

    for (int j=0; j<3; j++) {
        sum = hipCaddf(sum, vector[j]);
    }
    
    free(vector);

    return hipCrealf(sum);
}

*/
