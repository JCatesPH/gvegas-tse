#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

__device__
float heaviside(float x, float z)
{
    if (x < z)
    {
        return 0.f;
    }
    else
    {
        return 1.f;
    }
    
}


// ztest6
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(2*sizeof(hipFloatComplex));

    vector[0] = make_hipFloatComplex(rx[0], rx[1]);
    vector[1] = make_hipFloatComplex(rx[2], rx[3]);
    
    sum = hipCaddf(vector[0], vector[1]);
    
    free(vector);

    return hipCrealf(sum);
}


/*

// ztest5, dbltest
__device__
float func(float* rx, float wgt)
{
    double sum = 0;
    double *vector; 
    vector = (double*)malloc(4*sizeof(double));

    for (int j=0; j<4; j++) {
        vector[j] = rx[j];
    }
    
    for (int j=0; j<4; j++) {
        sum += vector[j];
    }

    free(vector);

    return (float)sum;
}
*/

/*

// ztest2/3/4
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex z1 = make_hipFloatComplex(rx[0], rx[1]);
    hipFloatComplex z2 = make_hipFloatComplex(rx[2], rx[3]);
    
    hipFloatComplex result;

    result = hipCdivf(z1, z2);

    return A * hipCrealf(result);
}

*/


/*

// ztest1
__device__
float func(float* rx, float wgt)
{
    hipFloatComplex sum = make_hipFloatComplex(0.f, 0.f);
    hipFloatComplex *vector;
    vector = (hipFloatComplex*)malloc(5*sizeof(hipFloatComplex));

    int i = 0;
    for (int j=0; j<10; j=j+2) {
        vector[i] = make_hipFloatComplex(rx[j], rx[j+1]);
        i++;
    }

    for (int j=0; j<5; j++) {
        sum = hipCaddf(sum, vector[j]);
    }
    
    free(vector);

    return hipCrealf(sum);
}

*/
