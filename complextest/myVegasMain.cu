#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>
#include <assert.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"




int main(int argc, char* argv[])
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n "ncall0" -i "itmx0" -a "nacc" -b "nBlockSize0" -d "ndim0"
   //
   // parameters:
   //   ncall = 1024*ncall0 is the amount of function calls
   //   itmx  = itmx0 is the maximum iterations for the algorithm
   //   acc   = nacc*0.00001f is the desired accuracy
   //   nBlockSize = nBlockSize0 is the size of the CUDA block
   //   ndim = ndim0 is the dimension of the integration space

   int ncall0 = 100;
   int itmx0 = 10;
   int nacc  = 1;
   int nBlockSize0 = 320;
   int ndim0 = 10;
   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:d:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
           case 'd':
             ndim0 = atoi(optarg);
             break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = (1 << ncall0)*1024;
   itmx = itmx0;
   acc = (float)nacc*0.000001f;
   nBlockSize = nBlockSize0;
   ndim = ndim0;

   assert(ndim <= ndim_max);

   mds = 1;

   ng = 0;
   npg = 0;

   /*-------- Setting integration limits ---------*/
   for (int i=0;i<ndim;i++) {
      xl[i] = 1.;
      xu[i] = 10.;
   }
 
    /*---------------------------------------------*/
//If nprn = 1 it prints the whole work, when nprn = 0, just the text in this code.
//If nprn = -1, we can get the grid update information.

  nprn = 1;
//  nprn = -1;
//  nprn = 0;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   myVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   //Print running times!
   std::cout<<"#==========================="<<std::endl;
   printf("# Function call time per iteration: %lf\n", timeVegasCallAndFill/(double)it);
   printf("# Refining time per iteration: %lf\n", timeVegasRefine/(double)it);
   std::cout<<"#==========================="<<std::endl;


    /* Instructions for performance measure
    char archivo[64];
    sprintf(archivo, "./datos/testtoolbox/prodexp.dat");
    FILE *f = fopen(archivo, "ab+");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    fprintf(f, "%d %d %.9lf %.9lf %lf\n", ndim0, ncall0, avgi, sd, timeVegasCallAndFill+timeVegasRefine);
    fclose(f);
    */
   return 0;
}
