#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

__device__
float func(double *rx, double wgt)
{
    float result;
    hipDoubleComplex  vkqt;
    hipDoubleComplex  vkqb;
    hipDoubleComplex  vkt;
    hipDoubleComplex  vkb;
    float N2 = (N - 1) / 2;

    vkqt = make_hipDoubleComplex(-V0 * (rx[2] + qy), -V0 * (rx[1] + qx));
    vkqb = hipConj(vkqt);

    vkt = make_hipDoubleComplex(-V0 * (rx[2]), -V0 * (rx[1]));
    vkb = hipConj(vkt);

    hipDoubleComplex *thekq, *phikq, *thek, *phik, *Grkq, *Grk, *Gakq, *Gak, *pvkqt, *pvkqb, *pvkt, *pvkb, *mp;

    thekq = (hipDoubleComplex*)malloc((N+1)*sizeof(hipDoubleComplex));
    phikq = (hipDoubleComplex*)malloc((N+2)*sizeof(hipDoubleComplex));

    thek = (hipDoubleComplex*)malloc((N+1)*sizeof(hipDoubleComplex));
    phik = (hipDoubleComplex*)malloc((N+2)*sizeof(hipDoubleComplex));

    Grkq = (hipDoubleComplex*)malloc(N*N*sizeof(hipDoubleComplex));
    Grk = (hipDoubleComplex*)malloc(N*N*sizeof(hipDoubleComplex));
    Gakq = (hipDoubleComplex*)malloc(N*N*sizeof(hipDoubleComplex));
    Gak = (hipDoubleComplex*)malloc(N*N*sizeof(hipDoubleComplex));

    pvkqt = (hipDoubleComplex*)malloc(N*sizeof(hipDoubleComplex));
    pvkqb = (hipDoubleComplex*)malloc(N*sizeof(hipDoubleComplex));
    pvkt = (hipDoubleComplex*)malloc(N*sizeof(hipDoubleComplex));
    pvkb = (hipDoubleComplex*)malloc(N*sizeof(hipDoubleComplex));
    mp = (hipDoubleComplex*)malloc(N*sizeof(hipDoubleComplex));

    hipDoubleComplex thzkq = make_hipDoubleComplex(1.f, 0.f);
    hipDoubleComplex thokq = make_hipDoubleComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2) * hOmg, Gamm);

    hipDoubleComplex thzk = make_hipDoubleComplex(1.f, 0.f);
    hipDoubleComplex thok = make_hipDoubleComplex(rx[0] - A * ((rx[1]) * (rx[1]) + (rx[2]) * (rx[2])) - V2 - (N2) * hOmg, Gamm);

    hipDoubleComplex phinpkq = make_hipDoubleComplex(1.f, 0.f);
    hipDoubleComplex phinkq = make_hipDoubleComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (N - 1)) * hOmg, Gamm);

    hipDoubleComplex phinpk = make_hipDoubleComplex(1.f, 0.f);
    hipDoubleComplex phink = make_hipDoubleComplex(rx[0] - A * ((rx[1]) * (rx[1]) + (rx[2]) * (rx[2])) - V2 - (N2 - (N - 1)) * hOmg, Gamm);

    thekq[0] = thzkq;
    thekq[1] = thokq;
    thek[0] = thzk;
    thek[1] = thok;

    phikq[0] = make_hipDoubleComplex(0.0, 0.0);
    phikq[N+1] = phinpkq;
    phikq[N] = phinkq;

    phik[0] = make_hipDoubleComplex(0.0, 0.0);
    phik[N+1] = phinpk;
    phik[N] = phink;

    hipDoubleComplex vnkqt = make_hipDoubleComplex(1.0, 0.0);
    pvkqt[0] = vnkqt;
    hipDoubleComplex vnkqb = make_hipDoubleComplex(1.0, 0.0);
    pvkqb[0] = vnkqb;
    hipDoubleComplex vnkt = make_hipDoubleComplex(1.0, 0.0);
    pvkt[0] = vnkt;
    hipDoubleComplex vnkb = make_hipDoubleComplex(1.0, 0.0);
    pvkb[0] = vnkb;

    hipDoubleComplex mn = make_hipDoubleComplex(1.0, 0.0);
    mp[0] = mn;

    hipDoubleComplex theskq;
    hipDoubleComplex phiskq;
    hipDoubleComplex thesk;
    hipDoubleComplex phisk;

    //for ss in range(2, N + 1):
    for(int ss=2; ss < N+1; ss++) {
        int s = N - ss + 1;
        int n = ss - 1;

        theskq = make_hipDoubleComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (ss - 1)) * hOmg, Gamm); 
        theskq = hipCsub(
            hipCmul(theskq, thokq), 
            hipCmul(vkqt, hipCmul(vkqb, thzkq))
        );
        thzkq = thokq;
        thokq = theskq;

        thekq[ss] = thokq;

        phiskq = make_hipDoubleComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (s - 1)) * hOmg, Gamm);
        phiskq = hipCsub(
            hipCmul(phiskq, phinkq), 
            hipCmul(vkqt, hipCmul(vkqb, phinpkq))
        );
        phinpkq = phinkq;
        phinkq = phiskq;

        phikq[s] = phinkq;

        thesk = make_hipDoubleComplex(rx[0] - A * (rx[1] * rx[1] + rx[2] * rx[2]) - V2 - (N2 - (ss - 1)) * hOmg, Gamm);
        thesk = hipCsub(
            hipCmul(theskq, thok), 
            hipCmul(vkt, hipCmul(vkb, thzk))
        );
        thzk = thok;
        thok = thesk;

        thek[ss] = thok;

        phisk = make_hipDoubleComplex(rx[0] - A * (rx[1] * rx[1] + rx[2] * rx[2]) - V2 - (N2 - (s - 1)) * hOmg, Gamm);
        phisk = hipCsub(
            hipCmul(phisk, phink), 
            hipCmul(vkt, hipCmul(vkb, phinpk))
        );
        phinpk = phink;
        phink = phisk;

        phik[s] = phink;

        vnkqt = hipCmul(vnkqt, vkqt);
        vnkt = hipCmul(vnkt, vkt);
        vnkqb = hipCmul(vnkqb, vkqb);
        vnkb = hipCmul(vnkb, vkb);

        pvkqt[n] = vnkqt;
        pvkt[n] = vnkt;

        pvkqb[n] = vnkqb;
        pvkb[n] = vnkb;

        mn = hipCmul(mn, make_hipDoubleComplex(-1.0, 0.0));
        mp[n] = mn;
    }

    //for m in range(0, N):
    for(int m=0; m<N; m++) {
        //for n in range(m, N):
        for(int n=m; n<N; n++) {
            //if m == n:
            if(m==n){
                //Grkq[m, n] = thekq[m] * phikq[n + 2] / thekq[N]
                Grkq[IDX2C(m,n,N)] = hipCdiv(
                    hipCmul(thekq[m], phikq[n + 2]), 
                    thekq[N]
                );
                //Gakq[IDX2C(m,n,N)] = complex(Grkq[IDX2C(m,n,N)].real, -Grkq[IDX2C(m,n,N)].imag);
                Gakq[IDX2C(m,n,N)] = hipConj(Grkq[IDX2C(m,n,N)]);

                //Grk[IDX2C(m,n,N)] = thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(m,n,N)] = hipCdiv(
                    hipCmul(thek[m], phik[n + 2]), 
                    thek[N]
                );
                //Gak[IDX2C(m,n,N)] = complex(Grk[IDX2C(m,n,N)].real, -Grk[IDX2C(m,n,N)].imag);
                Gak[IDX2C(m,n,N)] = hipConj(Grk[IDX2C(m,n,N)]);
            }
            //elif m < n:
            else if(m<n) {

                Grkq[IDX2C(m,n,N)] = hipCmul(
                    hipCmul(mp[n - m], pvkqt[n - m]),
                    hipCmul(
                        thekq[m], 
                        hipCdiv(phikq[n + 2], thekq[N])
                    )
                );
                //Grkq[IDX2C(n,m,N)] = mp[n - m] * pvkqb[n - m] * thekq[m] * phikq[n + 2] / thekq[N];
                Grkq[IDX2C(n,m,N)] = hipCmul(
                    hipCmul(mp[n - m], pvkqb[n - m]),
                    hipCmul(
                        thekq[m], 
                        hipCdiv(phikq[n + 2], thekq[N])
                    )
                );
                //Gakq[IDX2C(m,n,N)] = complex(Grkq[IDX2C(n,m,N)].real, -Grkq[IDX2C(n,m,N)].imag);
                //Gakq[IDX2C(n,m,N)] = complex(Grkq[IDX2C(m,n,N)].real, -Grkq[IDX2C(m,n,N)].imag);
                Gakq[IDX2C(m,n,N)] = hipConj(Grkq[IDX2C(n,m,N)]);
                Gakq[IDX2C(n,m,N)] = hipConj(Grkq[IDX2C(m,n,N)]);

                //Grk[IDX2C(m,n,N)] = mp[n - m] * pvkt[n - m] * thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(m,n,N)] = hipCmul(
                    hipCmul(mp[n - m], pvkt[n - m]),
                    hipCmul(
                        thek[m], 
                        hipCdiv(phik[n + 2], thek[N])
                    )
                );
                //Grk[IDX2C(n,m,N)] = mp[n - m] * pvkb[n - m] * thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(n,m,N)] = hipCmul(
                    hipCmul(mp[n - m], pvkb[n - m]),
                    hipCmul(
                        thek[m], 
                        hipCdiv(phik[n + 2], thek[N])
                    )
                );
                //Gak[IDX2C(m,n,N)] = complex(Grk[IDX2C(n,m,N)].real, -Grk[IDX2C(n,m,N)].imag);
                //Gak[IDX2C(n,m,N)] = complex(Grk[IDX2C(m,n,N)].real, -Grk[IDX2C(m,n,N)].imag);
                Gak[IDX2C(m,n,N)] = hipConj(Grk[IDX2C(n,m,N)]);
                Gak[IDX2C(n,m,N)] = hipConj(Grk[IDX2C(m,n,N)]);
            }
        }
    }

    hipDoubleComplex chi1f = make_hipDoubleComplex(0, 0);
//
    //for l in range(0, N):
    for(int l=0; l<N; l++) {
        //if (mu - x[0] + ((N - 1) / 2 - l) * hOmg) < 0:
        if((mu - rx[0] + (N2 - l) * hOmg) >= 0) {
            //for m in range(0, N):
            for(int m=0; m<N; m++) {
                //for n in range(0, N):
                for(int n=0; n<N; n++) {
                    //chi1f += Grkq[IDX2C(m,n,N)] * Grk[IDX2C(n,l,N)] * Gak[IDX2C(l,m,N)] + Grkq[IDX2C(n,l,N)] * Gakq[IDX2C(l,m,N)] * Gak[IDX2C(m,n,N)];
                    chi1f = hipCadd(
                        chi1f,
                        hipCadd(
                            hipCmul(Grkq[IDX2C(m,n,N)], hipCmul(Grk[IDX2C(n,l,N)], Gak[IDX2C(l,m,N)])),
                            hipCmul(Grkq[IDX2C(n,l,N)], hipCmul(Gakq[IDX2C(l,m,N)], Gak[IDX2C(m,n,N)]))
                        )
                    );
                }
            }
        }
    }

    result = Fac*hipCreal(chi1f);

    free(thekq); 
    free(phikq); 
    free(thek); 
    free(phik); 
    free(Grkq); 
    free(Grk); 
    free(Gakq); 
    free(Gak); 
    free(pvkqt); 
    free(pvkqb); 
    free(pvkt); 
    free(pvkb); 
    free(mp);

    return result;
}

