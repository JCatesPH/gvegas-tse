#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

__device__
float func(double *rx, double wgt)
{
    float result;
    hipFloatComplex  vkqt;
    hipFloatComplex  vkqb;
    hipFloatComplex  vkt;
    hipFloatComplex  vkb;
    float N2 = (N - 1) / 2;

    vkqt = make_hipFloatComplex(-V0 * (rx[2] + qy), -V0 * (rx[1] + qx));
    vkqb = hipConjf(vkqt);

    vkt = make_hipFloatComplex(-V0 * (rx[2]), -V0 * (rx[1]));
    vkb = hipConjf(vkt);

    hipFloatComplex *thekq, *phikq, *thek, *phik, *Grkq, *Grk, *Gakq, *Gak, *pvkqt, *pvkqb, *pvkt, *pvkb, *mp;

    thekq = (hipFloatComplex*)malloc((N+1)*sizeof(hipFloatComplex));
    phikq = (hipFloatComplex*)malloc((N+2)*sizeof(hipFloatComplex));

    thek = (hipFloatComplex*)malloc((N+1)*sizeof(hipFloatComplex));
    phik = (hipFloatComplex*)malloc((N+2)*sizeof(hipFloatComplex));

    Grkq = (hipFloatComplex*)malloc(N*N*sizeof(hipFloatComplex));
    Grk = (hipFloatComplex*)malloc(N*N*sizeof(hipFloatComplex));
    Gakq = (hipFloatComplex*)malloc(N*N*sizeof(hipFloatComplex));
    Gak = (hipFloatComplex*)malloc(N*N*sizeof(hipFloatComplex));

    pvkqt = (hipFloatComplex*)malloc(N*sizeof(hipFloatComplex));
    pvkqb = (hipFloatComplex*)malloc(N*sizeof(hipFloatComplex));
    pvkt = (hipFloatComplex*)malloc(N*sizeof(hipFloatComplex));
    pvkb = (hipFloatComplex*)malloc(N*sizeof(hipFloatComplex));
    mp = (hipFloatComplex*)malloc(N*sizeof(hipFloatComplex));

    hipFloatComplex thzkq = make_hipFloatComplex(1.f, 0.f);
    hipFloatComplex thokq = make_hipFloatComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2) * hOmg, Gamm);

    hipFloatComplex thzk = make_hipFloatComplex(1.f, 0.f);
    hipFloatComplex thok = make_hipFloatComplex(rx[0] - A * ((rx[1]) * (rx[1]) + (rx[2]) * (rx[2])) - V2 - (N2) * hOmg, Gamm);

    hipFloatComplex phinpkq = make_hipFloatComplex(1.f, 0.f);
    hipFloatComplex phinkq = make_hipFloatComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (N - 1)) * hOmg, Gamm);

    hipFloatComplex phinpk = make_hipFloatComplex(1.f, 0.f);
    hipFloatComplex phink = make_hipFloatComplex(rx[0] - A * ((rx[1]) * (rx[1]) + (rx[2]) * (rx[2])) - V2 - (N2 - (N - 1)) * hOmg, Gamm);

    thekq[0] = thzkq;
    thekq[1] = thokq;
    thek[0] = thzk;
    thek[1] = thok;

    phikq[0] = make_hipFloatComplex(0.0, 0.0);
    phikq[N+1] = phinpkq;
    phikq[N] = phinkq;

    phik[0] = make_hipFloatComplex(0.0, 0.0);
    phik[N+1] = phinpk;
    phik[N] = phink;

    hipFloatComplex vnkqt = make_hipFloatComplex(1.0, 0.0);
    pvkqt[0] = vnkqt;
    hipFloatComplex vnkqb = make_hipFloatComplex(1.0, 0.0);
    pvkqb[0] = vnkqb;
    hipFloatComplex vnkt = make_hipFloatComplex(1.0, 0.0);
    pvkt[0] = vnkt;
    hipFloatComplex vnkb = make_hipFloatComplex(1.0, 0.0);
    pvkb[0] = vnkb;

    hipFloatComplex mn = make_hipFloatComplex(1.0, 0.0);
    mp[0] = mn;

    //for ss in range(2, N + 1):
    for(int ss=2; ss < N+1; ss++) {
        int s = N - ss + 1;
        int n = ss - 1;

        hipFloatComplex theskq = make_hipFloatComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (ss - 1)) * hOmg, Gamm); 
        theskq = hipCsubf(
            hipCmulf(theskq, thokq), 
            hipCmulf(vkqt, hipCmulf(vkqb, thzkq))
        );
        thzkq = thokq;
        thokq = theskq;

        thekq[ss] = thokq;

        hipFloatComplex phiskq = make_hipFloatComplex(rx[0] - A * ((rx[1] + qx) * (rx[1] + qx) + (rx[2] + qy) * (rx[2] + qy)) - V2 - (N2 - (s - 1)) * hOmg, Gamm);
        phiskq = hipCsubf(
            hipCmulf(phiskq, phinkq), 
            hipCmulf(vkqt, hipCmulf(vkqb, phinpkq))
        );
        phinpkq = phinkq;
        phinkq = phiskq;

        phikq[s] = phinkq;

        hipFloatComplex thesk = make_hipFloatComplex(rx[0] - A * (rx[1] * rx[1] + rx[2] * rx[2]) - V2 - (N2 - (ss - 1)) * hOmg, Gamm);
        thesk = hipCsubf(
            hipCmulf(theskq, thok), 
            hipCmulf(vkt, hipCmulf(vkb, thzk))
        );
        thzk = thok;
        thok = thesk;

        thek[ss] = thok;

        hipFloatComplex phisk = make_hipFloatComplex(rx[0] - A * (rx[1] * rx[1] + rx[2] * rx[2]) - V2 - (N2 - (s - 1)) * hOmg, Gamm);
        phisk = hipCsubf(
            hipCmulf(phisk, phink), 
            hipCmulf(vkt, hipCmulf(vkb, phinpk))
        );
        phinpk = phink;
        phink = phisk;

        phik[s] = phink;

        vnkqt = hipCmulf(vnkqt, vkqt);
        vnkt = hipCmulf(vnkt, vkt);
        vnkqb = hipCmulf(vnkqb, vkqb);
        vnkb = hipCmulf(vnkb, vkb);

        pvkqt[n] = vnkqt;
        pvkt[n] = vnkt;

        pvkqb[n] = vnkqb;
        pvkb[n] = vnkb;

        mn = hipCmulf(mn, make_hipFloatComplex(-1.0, 0.0));
        mp[n] = mn;
    }

    //for m in range(0, N):
    for(int m=0; m<N; m++) {
        //for n in range(m, N):
        for(int n=m; n<N; n++) {
            //if m == n:
            if(m==n){
                //Grkq[m, n] = thekq[m] * phikq[n + 2] / thekq[N]
                Grkq[IDX2C(m,n,N)] = hipCdivf(
                    hipCmulf(thekq[m], phikq[n + 2]), 
                    thekq[N]
                );
                //Gakq[IDX2C(m,n,N)] = complex(Grkq[IDX2C(m,n,N)].real, -Grkq[IDX2C(m,n,N)].imag);
                Gakq[IDX2C(m,n,N)] = hipConjf(Grkq[IDX2C(m,n,N)]);

                //Grk[IDX2C(m,n,N)] = thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(m,n,N)] = hipCdivf(
                    hipCmulf(thek[m], phik[n + 2]), 
                    thek[N]
                );
                //Gak[IDX2C(m,n,N)] = complex(Grk[IDX2C(m,n,N)].real, -Grk[IDX2C(m,n,N)].imag);
                Gak[IDX2C(m,n,N)] = hipConjf(Grk[IDX2C(m,n,N)]);
            }
            //elif m < n:
            else if(m<n) {

                Grkq[IDX2C(m,n,N)] = hipCmulf(
                    hipCmulf(mp[n - m], pvkqt[n - m]),
                    hipCmulf(
                        thekq[m], 
                        hipCdivf(phikq[n + 2], thekq[N])
                    )
                );
                //Grkq[IDX2C(n,m,N)] = mp[n - m] * pvkqb[n - m] * thekq[m] * phikq[n + 2] / thekq[N];
                Grkq[IDX2C(n,m,N)] = hipCmulf(
                    hipCmulf(mp[n - m], pvkqb[n - m]),
                    hipCmulf(
                        thekq[m], 
                        hipCdivf(phikq[n + 2], thekq[N])
                    )
                );
                //Gakq[IDX2C(m,n,N)] = complex(Grkq[IDX2C(n,m,N)].real, -Grkq[IDX2C(n,m,N)].imag);
                //Gakq[IDX2C(n,m,N)] = complex(Grkq[IDX2C(m,n,N)].real, -Grkq[IDX2C(m,n,N)].imag);
                Gakq[IDX2C(m,n,N)] = hipConjf(Grkq[IDX2C(n,m,N)]);
                Gakq[IDX2C(n,m,N)] = hipConjf(Grkq[IDX2C(m,n,N)]);

                //Grk[IDX2C(m,n,N)] = mp[n - m] * pvkt[n - m] * thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(m,n,N)] = hipCmulf(
                    hipCmulf(mp[n - m], pvkt[n - m]),
                    hipCmulf(
                        thek[m], 
                        hipCdivf(phik[n + 2], thek[N])
                    )
                );
                //Grk[IDX2C(n,m,N)] = mp[n - m] * pvkb[n - m] * thek[m] * phik[n + 2] / thek[N];
                Grk[IDX2C(n,m,N)] = hipCmulf(
                    hipCmulf(mp[n - m], pvkb[n - m]),
                    hipCmulf(
                        thek[m], 
                        hipCdivf(phik[n + 2], thek[N])
                    )
                );
                //Gak[IDX2C(m,n,N)] = complex(Grk[IDX2C(n,m,N)].real, -Grk[IDX2C(n,m,N)].imag);
                //Gak[IDX2C(n,m,N)] = complex(Grk[IDX2C(m,n,N)].real, -Grk[IDX2C(m,n,N)].imag);
                Gak[IDX2C(m,n,N)] = hipConjf(Grk[IDX2C(n,m,N)]);
                Gak[IDX2C(n,m,N)] = hipConjf(Grk[IDX2C(m,n,N)]);
            }
        }
    }

    hipFloatComplex chi1f = make_hipFloatComplex(0, 0);
//
    //for l in range(0, N):
    for(int l=0; l<N; l++) {
        //if (mu - x[0] + ((N - 1) / 2 - l) * hOmg) < 0:
        if((mu - rx[0] + (N2 - l) * hOmg) >= 0) {
            //for m in range(0, N):
            for(int m=0; m<N; m++) {
                //for n in range(0, N):
                for(int n=0; n<N; n++) {
                    //chi1f += Grkq[IDX2C(m,n,N)] * Grk[IDX2C(n,l,N)] * Gak[IDX2C(l,m,N)] + Grkq[IDX2C(n,l,N)] * Gakq[IDX2C(l,m,N)] * Gak[IDX2C(m,n,N)];
                    chi1f = hipCaddf(
                        chi1f,
                        hipCaddf(
                            hipCmulf(Grkq[IDX2C(m,n,N)], hipCmulf(Grk[IDX2C(n,l,N)], Gak[IDX2C(l,m,N)])),
                            hipCmulf(Grkq[IDX2C(n,l,N)], hipCmulf(Gakq[IDX2C(l,m,N)], Gak[IDX2C(m,n,N)]))
                        )
                    );
                }
            }
        }
    }

    result = Fac*hipCrealf(chi1f);

    free(thekq); 
    free(phikq); 
    free(thek); 
    free(phik); 
    free(Grkq); 
    free(Grk); 
    free(Gakq); 
    free(Gak); 
    free(pvkqt); 
    free(pvkqb); 
    free(pvkt); 
    free(pvkb); 
    free(mp);

    return result;
}

