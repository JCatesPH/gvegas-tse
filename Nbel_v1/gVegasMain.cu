#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <sstream>

#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
//#include <cutil_inline.h>
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "getrusage_sec.h"
#include "kernels.h"


int main(int argc, char** argv)
{

   //
   // program interface:
   //   program -ncall="ncall0" -itmx="itmx0" -acc="acc0" -b="nBlockSize0"
   //
   // parameters:
   //   ncall0 = "exxny"
   //   ncall = y*10^xx
   //   itmx  = itmx0
   //   acc   = 0.01*acc0
   //   nBlockSize = nBlockSize0
   //

   //------------------
   //  Initialization
   //------------------

   int itmx0 = 10;
   int nBlockSize0 = 256;
   int GPUdevice = 0;

   float acc0 = 0.0001f;

   ncall = 1024*32;
   itmx = itmx0;
   acc = 0.01*acc0;
   nBlockSize = nBlockSize0;

   hipSetDevice(GPUdevice);

   mds = 1;
   ndim = 3;
   
   ng = 0;
   npg = 0;

   /*-------- Setting integration limits ---------*/
  /* for (int i=0;i<ndim;i++) {
    xl[i] = 0.;
    xu[i] = 1.;
  }*/

  // Based on original description of problem. 
  //"The integrand is Ds(kx,ky,qx,qy)/(2*pi)^3, and the limits of integration are kx=[-pi/a,pi/a],ky=[-pi/a,pi/a] , qx=[-pi/a,pi/a] and qy=[-pi/a,pi/a]."
  //"For qx and qy it is more efficient to use qx=[0.001,pi/a] and qy=0, because of the symmetry of the problem. kx and ky should be as we said before kx=[-pi/a,pi/a],ky=[-pi/a,pi/a]."

  xl[0] = -0.25;
  xu[0] = 0.25;

  xl[1] = -3.14159265358979 / 4.0; // kxi
  xu[1] = 3.14159265358979  / 4.0; // kxf

  xl[2] = -3.14159265358979 / 4.0; // kyi
  xu[2] = 3.14159265358979f / 4.0; // kyf
   
   nprn = 1;
   //   nprn = -1;
   /*----------------------------------------------*/

   double startTotal, endTotal, timeTotal;
   timeTotal = 0.;
   startTotal = getrusage_usec();

   timeVegasCall = 0.;
   timeVegasMove = 0.;
   timeVegasFill = 0.;
   timeVegasRefine = 0.;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   gVegas(avgi, sd, chi2a);

   endTotal = getrusage_usec();
   timeTotal = endTotal - startTotal;

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<std::setw(10)<<std::setprecision(6)<<std::endl;
   std::cout<<"#============================="<<std::endl;
   std::cout<<"# No. of Thread Block Size  : "<<nBlockSize<<std::endl;
   std::cout<<"#============================="<<std::endl;
   std::cout<<"# No. of dimensions         : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter  : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations    : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy          : "<<acc<<std::endl;
   std::cout<<"#============================="<<std::endl;
   std::cout<<std::scientific;
   std::cout<<std::left<<std::setfill(' ');
   std::cout<<"# Result                    : "
            <<std::setw(12)<<std::setprecision(5)<<avgi<<" +- "
            <<std::setw(12)<<std::setprecision(5)<<sd<<" ( "
            <<std::setw(7)<<std::setprecision(4)
            <<std::fixed<<100.*sd/avgi<<"%)"<<std::endl;
   std::cout<<std::fixed;
   std::cout<<"# Chisquare                 : "<<std::setprecision(4)
            <<chi2a<<std::endl;
   std::cout<<"#============================="<<std::endl;
   std::cout<<std::right;
   std::cout<<"# Total Execution Time(sec) : "
            <<std::setw(10)<<std::setprecision(4)<<timeTotal<<std::endl;
   std::cout<<"#============================="<<std::endl;
   std::cout<<"# Time for func calls (sec) : "
            <<std::setw(10)<<std::setprecision(4)<<timeVegasCall
            <<" ( "<<std::setw(5)<<std::setprecision(2)
            <<100.*timeVegasCall/timeTotal<<"%)"<<std::endl;
   std::cout<<"# Time for data transf (sec): "
            <<std::setw(10)<<std::setprecision(4)<<timeVegasMove
            <<" ( "<<std::setw(5)<<std::setprecision(2)
            <<100.*timeVegasMove/timeTotal<<"%)"<<std::endl;
   std::cout<<"# Time for data fill (sec)  : "
            <<std::setw(10)<<std::setprecision(4)<<timeVegasFill
            <<" ( "<<std::setw(5)<<std::setprecision(2)
            <<100.*timeVegasFill/timeTotal<<"%)"<<std::endl;
   std::cout<<"# Time for grid refine (sec): "
            <<std::setw(10)<<std::setprecision(4)<<timeVegasRefine
            <<" ( "<<std::setw(5)<<std::setprecision(2)
            <<100.*timeVegasRefine/timeTotal<<"%)"<<std::endl;
   std::cout<<"#============================="<<std::endl;

   hipDeviceReset();

   return 0;
}
