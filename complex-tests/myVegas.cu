#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <omp.h>
#include <assert.h>

#include "hip/hip_runtime_api.h"

#include "vegas.h"
#include "vegasconst.h"
#include "kernels.h"

#include "gvegas.h"

void myVegas(double& avgi, double& sd, double& chi2a)
{

   for (int j=0;j<ndim;j++) {
      xi[j][0] = 1.f;
      for (int i = 1; i < nd_max; i++) {
        xi[j][i] = 0.f;
      }
   }
   /*
   Original code doesn't account for the rest of xi, just assumes that when it
   declares the array, the rest will be set to zeroes, and that will not always
   happen.
   */
   // entry vegas1

   it = 0;

   // entry vegas2
   nd = nd_max;
   ng = 1;

   npg = 0;
   //std::cout<<"mds = "<<mds<<std::endl;
   if (mds!=0) {

      std::cout<<"ncall, ndim = "<<ncall<<", "<<ndim<<std::endl;
      ng = (int)pow((0.5*(double)ncall),1./(double)ndim);
      mds = 1;
      //      printf("ng = %d\n",ng);

      if (2*ng>=nd_max) {
         mds = -1;
         npg = ng/nd_max+1;
         nd = ng/npg;
         ng = npg*nd;
      }

   }
   std::cout<<"mds = "<<mds<<std::endl;
   //assert(mds == 1);

   //std::cout<<"ng = "<<ng<<std::endl;
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &ndim, sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ng),   &ng,   sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_nd),   &nd,   sizeof(int)));
   hipDeviceSynchronize(); // wait for synchronize

   nCubes = (unsigned)(pow(ng,ndim));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_nCubes), &nCubes, sizeof(nCubes)));
   hipDeviceSynchronize(); // wait for synchronize

   npg = ncall/nCubes;
   if (npg<2) npg = 2;
   calls = (double)(npg*nCubes);

   unsigned nCubeNpg = nCubes*npg;

   //std::cout<<"nCubes= "<<nCubes<<std::endl;
   //std::cout<<"nCubeNpg= "<<nCubeNpg<<std::endl;

   if (nprn!=0) {
      // tsi = sqrt(tsi);
      std::cout<<std::endl;
      std::cout<<" << vegas internal parameters >>"<<std::endl;
      std::cout<<"            ng: "<<std::setw(5)<<ng<<std::endl;
      std::cout<<"            nd: "<<std::setw(5)<<nd<<std::endl;
      std::cout<<"           npg: "<<std::setw(5)<<npg<<std::endl;
      std::cout<<"        nCubes: "<<std::setw(12)<<nCubes<<std::endl;
      std::cout<<"    nCubes*npg: "<<std::setw(12)<<nCubeNpg<<std::endl;
   }

   dxg = 1.f/(float)ng;
   double dnpg = (double)npg;
   double dv2g = calls*calls*pow(dxg,ndim)*pow(dxg,ndim)/(dnpg*dnpg*(dnpg-1.));
   xnd = (float)nd;
   dxg *= xnd;
   xjac = 1.f/(float)calls;
   for (int j=0;j<ndim;j++) {
      dx[j] = xu[j]-xl[j];
      xjac *= dx[j];
   }

   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_npg),  &npg,  sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xjac), &xjac, sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_dxg),  &dxg,  sizeof(float)));
   hipDeviceSynchronize(); // wait for synchronize

   //----------------------------------
   //  Set parameters in the integrand.
   //----------------------------------
   float mu_h     = 0.1f;
   float hOmg_h   = 0.3f;
   float a_h      = 3.6f;
   float A_h      = 4.f;
   float rati_h   = 0.1;
   float eE0_h    = rati * (hOmg * hOmg) / (2 * sqrt(A * mu));
   float Gamm_h   = 0.003;
   float KT_h     = 1e-6;
   float shift_h  = A * (eE0 / hOmg) * (eE0 / hOmg);
   float Gammsq_h = Gamm * Gamm;
   int   N_h      = 3;

   // Move the parameters to the GPU memory.
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mu),          &mu_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(hOmg),      &hOmg_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(a),            &a_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(A),            &A_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(rati),      &rati_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(eE0),        &eE0_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Gamm),      &Gamm_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(KT),          &KT_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(shift),    &shift_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Gammsq),  &Gammsq_h,  sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(N),            &N_h,  sizeof(int)));
   //----------------------------------

   ndo = 1;

   if (nd!=ndo) {

      double rc = (double)ndo/xnd;

      for (int j=0;j<ndim;j++) {

         int k = -1;
         double xn = 0.;
         double dr = 0.;
         int i = k;
         k++;
         dr += 1.;
         double xo = xn;
         xn = xi[j][k];
         //         printf("xn = %g\n",xn);
         while (i<nd-1) {

            while (dr<=rc) {
               k++;
               dr += 1.;
               xo = xn;
               xn = xi[j][k];
               //printf("xn = %g\n",xn);
            }
            i++;
            dr -= rc;
            xin[i] = xn - (xn-xo)*dr;
         }

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      ndo = nd;

   }

   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xl), xl, sizeof(xl)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_dx), dx, sizeof(dx)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
   hipDeviceSynchronize(); // wait for synchronize

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << input parameters for vegas >>"<<std::endl;
      std::cout<<"     ndim ="<<std::setw(3)<<ndim
               <<"   ncall ="<<std::setw(10)<<(int)calls<<std::endl;
      std::cout<<"     it   =  0"
               <<"   itmx ="<<std::setw(5)<<itmx<<std::endl;
      std::cout<<"     acc  = "<<std::fixed
               <<std::setw(9)<<std::setprecision(3)<<acc<<std::endl;
      std::cout<<"     mds  ="<<std::setw(3)<<mds
               <<"   nd = "<<std::setw(4)<<nd<<std::endl;
      for (int j=0;j<ndim;j++) {
         std::cout<<"    (xl,xu)= ( "<<std::setw(6)<<std::fixed
                  <<xl[j]<<" , "<<xu[j]<<" )"<<std::endl;
      }

   }

   // entry vegas3

   it = 0;
   si = 0.0f;
   si2 = 0.0f;
   swgt = 0.0f;
   schi = 0.0f;
   //   int iflag;
   // main integration loop

   //   std::cout<<"nBlockSize = "<<nBlockSize<<std::endl;
   //--------------------------
   //  Set up kernel variables
   //--------------------------
   //const int nGridSizeMax =  65535; //Original - Maximum size of grid in X for Fermi.
   const int nGridSizeMax = 1<<31 - 1; //This should be the one for current architectures.
   float hd[ndim_max][nd_max];

   dim3 ThBk(nBlockSize);

   int nGridSizeX, nGridSizeY;
   int nBlockTot = (nCubes-1)/nBlockSize+1;
   //std::cout<<"nBlockTot = "<<nBlockTot<<std::endl;
   nGridSizeY = (nBlockTot-1)/nGridSizeMax+1;
   nGridSizeX = (nBlockTot-1)/nGridSizeY+1;
   //std::cout<<"nGridSize (x,y) = "<<nGridSizeX<<", "<<nGridSizeY<<std::endl;
   dim3 BkGd(nGridSizeX, nGridSizeY);
   
   // Get a good grid for initzero()
   dim3 InitZeroTh(ndim,nd);

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << kernel parameters for CUDA >>"<<std::endl;
      std::cout<<"       Block size           ="<<std::setw(7)<<ThBk.x<<std::endl;
      std::cout<<"       Grid size            ="<<std::setw(7)<<BkGd.x
               <<" x "<<BkGd.y<<std::endl;
      int nThreadsTot = ThBk.x*BkGd.x*BkGd.y;
      std::cout<<"     Actual Number of calls ="<<std::setw(12)
               <<nThreadsTot*npg<<std::endl;
      std::cout<<"   Required Number of calls ="<<std::setw(12)
               <<nCubeNpg<<" ( "<<std::setw(6)<<std::setprecision(2)
               <<100.*(double)nCubeNpg/(double)(nThreadsTot*npg)<<"%)"<<std::endl;
      std::cout<<std::endl;
   }

   //By using the new GPU kernel we eliminate the need to move big stuff from
   //GPU to CPU and the need to run the Fill part.
   double startVegasCallAndFill, endVegasCallAndFill;
   double startVegasRefine, endVegasRefine;

   initzero<<<1,InitZeroTh>>>();
   getLastCudaError("initzero error");


   do {

      it++;

//      std::cout<<"call gVegasCallFunc: it = "<<it<<std::endl;
      startVegasCallAndFill = omp_get_wtime();

      // Initialize all values to zero, need to make a grid good enough to make everything faster...
      //initzero<<<1, 1>>>();
      // Now CallFilla will need a number of threads equal to the amount of cubes!
      myVegasCallFilla<<<BkGd, ThBk>>>(mds);
      getLastCudaError("myVegasCallFilla error");
      hipDeviceSynchronize(); // wait for synchronize
      checkCudaErrors(hipMemcpyFromSymbol(&ti, HIP_SYMBOL(doubleti), sizeof(double)));
      checkCudaErrors(hipMemcpyFromSymbol(&tsi, HIP_SYMBOL(doubletsi), sizeof(double)));
      checkCudaErrors(hipMemcpyFromSymbol(&hd, HIP_SYMBOL(d), sizeof(d)));
      //checkCudaErrors(hipMemcpyFromSymbol(&hd, HIP_SYMBOL(d), ndim_max*nd_max*sizeof(float)));

      endVegasCallAndFill = omp_get_wtime();
      timeVegasCallAndFill += endVegasCallAndFill-startVegasCallAndFill;

      //Initialize to zero before starting CPU computations to do everything at the same time
      initzero<<<1,InitZeroTh>>>();
      getLastCudaError("initzero error");

      tsi *= dv2g;
      double ti2 = (double)ti*(double)ti;
      double wgt = ti2/(double)tsi;
      si += ti*wgt;
      si2 += ti2;
      swgt += wgt;
      schi += ti2*wgt;
      avgi = si/swgt;
      sd = swgt*it/si2;
      chi2a = 0.;
      if (it>1) chi2a = sd*(schi/swgt-avgi*avgi)/((double)it-1.);
      sd = sqrt(1./sd);

      if (nprn!=0) {
         tsi = sqrt(tsi);
         std::cout<<std::endl;
         std::cout<<" << integration by vegas >>"<<std::endl;
         std::cout<<"     iteration no. "<<std::setw(4)<<it
                  <<"   integral=  "<<ti<<std::endl;
         std::cout<<"                          std dev  = "<<tsi<<std::endl;
         std::cout<<"     accumulated results: integral = "<<avgi<<std::endl;
         std::cout<<"                          std dev  = "<<sd<<std::endl;
	 if (it > 1) {
            std::cout<<"                          chi**2 per it'n = "
                     <<std::setw(10)<<std::setprecision(6)<<chi2a<<std::endl;
         }
         if (nprn<0) {
            for (int j=0;j<ndim;j++) {
               std::cout<<"   == data for axis "
                        <<std::setw(2)<<j<<" --"<<std::endl;
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce"<<std::endl;

               for (int i=0;i<nd;i+=3) {
                  std::cout<<std::setw(6)<<std::setprecision(6)<<std::setfill(' ')
                           <<xi[j][i]<<" "<<hd[j][i]<<" "<<hd[j][i];
                  std::cout<<std::setw(6)<<std::setprecision(4)
                           <<xi[j][i+1]<<" "<<hd[j][i+1]<<" "<<hd[j][i+1];
                  std::cout<<std::setw(6)<<std::setprecision(4)
                           <<xi[j][i+2]<<" "<<hd[j][i+2]<<" "<<hd[j][i+2]
                           <<std::endl;
                           }

            }
         }
      }

      // refine grid

      startVegasRefine = omp_get_wtime();

      /*
      for (int ii=0;ii<ndim;ii++) {
         for (int jj=0;jj<nd;jj++) {
            std::cout<<"d["<<ii<<"]["<<jj<<"] = "<<std::scientific
                     <<d[ii][jj]<<std::endl;
         }
      }
      */

      double r[nd_max];
      double dt[ndim_max];
      for (int j=0;j<ndim;j++) {
         double xo = hd[j][0];
         double xn = hd[j][1];
         hd[j][0] = 0.5*(xo+xn);
         dt[j] = hd[j][0];
         for (int i=1;i<nd-1;i++) {
            hd[j][i] = xo+xn;
            xo = xn;
            xn = hd[j][i+1];
            hd[j][i] = (hd[j][i]+xn)/3.;
            dt[j] += hd[j][i];
         }
         hd[j][nd-1] = 0.5*(xn+xo);
         dt[j] += hd[j][nd-1];
      }

      for (int j=0;j<ndim;j++) {
         double rc = 0.;
         for (int i=0;i<nd;i++) {
            r[i] = 0.;
            if (hd[j][i]>0.) {
               double xo = dt[j]/hd[j][i];
               if (!isinf(xo))
                  r[i] = pow(((xo-1.)/xo/log(xo)),alph);
            }
            rc += r[i];
         }
         rc /= xnd;
         int k = -1;
         double xn = 0.;
         double dr = xn;
         int i = k;
         k++;
         dr += r[k];
         double xo = xn;
         xn = xi[j][k];

         do {

            while (dr<=rc) {
               k++;
               dr += r[k];
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn-(xn-xo)*dr/r[k];

         } while (i<nd-2);

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
      hipDeviceSynchronize(); // wait for synchronize

      endVegasRefine = omp_get_wtime();
      timeVegasRefine += endVegasRefine-startVegasRefine;

//      std::cout<<"The end of main loop: it, sd/avgi = "<<it<<", "
//               <<sd/fabs(avgi)<<std::endl;

   } while (it<itmx && acc*fabs(avgi)<sd);

}
