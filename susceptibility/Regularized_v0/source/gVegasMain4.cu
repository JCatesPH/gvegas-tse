#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>

#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
//#include <cutil_inline.h>
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "getrusage_sec.h"
#include "kernels.h"


int main(int argc, char** argv)
{

  //
  // program interface:
  //   program -ncall="ncall0" -itmx="itmx0" -acc="acc0" -b="nBlockSize0"
  //
  // parameters:
  //   ncall0 = "exxny"
  //   ncall = y*10^xx
  //   itmx  = itmx0
  //   acc   = 0.01*acc0
  //   nBlockSize = nBlockSize0
  //

  //------------------
  //  Initialization
  //------------------

  int itmx0 = 100;
  int nBlockSize0 = 256;
  int GPUdevice = 0;

  float acc0 = 0.0001f;

  ncall = 1024*1000;
  itmx = itmx0;
  acc = acc0;
  nBlockSize = nBlockSize0;

  hipSetDevice(GPUdevice);

  mds = 1;
  ndim = 3;
  
  ng = 0;
  npg = 0;

   /*-------- Setting integration limits ---------*/
  /* for (int i=0;i<ndim;i++) {
    xl[i] = 0.;
    xu[i] = 1.;
  }*/

  // Based on original description of problem. 
  //"The integrand is Ds(kx,ky,qx,qy)/(2*pi)^3, and the limits of integration are kx=[-pi/a,pi/a],ky=[-pi/a,pi/a] , qx=[-pi/a,pi/a] and qy=[-pi/a,pi/a]."
  //"For qx and qy it is more efficient to use qx=[0.001,pi/a] and qy=0, because of the symmetry of the problem. kx and ky should be as we said before kx=[-pi/a,pi/a],ky=[-pi/a,pi/a]."

  xl[0] = -0.15;
  xu[0] = 0.15;

  xl[1] = -3.14159265358979 / 3.56; // kxi
  xu[1] = 3.14159265358979  / 3.56; // kxf

  xl[2] = -3.14159265358979 / 3.56; // kyi
  xu[2] = 3.14159265358979f / 3.56; // kyf

  //----------------------------------
  //  Set parameters in the integrand.
  //----------------------------------
  mu_h     = 0.115f;
  hOmg_h   = 0.3f;
  a_h      = 3.56f;
  t0_h      = 4.f;
  eA0a_h   = 0.35f;
  Gamm_h   = 0.003f;
  j0_h     = 0.969608676323187;
  j1_h     = 0.172333955218708;
  Gammsq_h = Gamm_h * Gamm_h;
  N_h      = 7;
  N2_h     = (N_h - 1) / 2;

  V0_h     = 2 * j0_h;
  V1_h     = j1_h;
  Fac_h    = -(a_h * a_h * Gamm_h / (PI*PI));
  qx_h     = 0.1; //0.01f + (PI / a_h) * 30.f / 50.f;
  qy_h     = 0.f;
   
  nprn = 1;
  //   nprn = -1;
  /*----------------------------------------------*/
  int num_qx = 101;
  double timeAllpts[num_qx];
  double resultsAllpts[num_qx];
  double errorAllpts[num_qx];

  for(int i=75; i < num_qx; i++) {
        qx_h = i * PI / (a_h * num_qx);

        std::cout<<std::endl<<"#============================="<<std::endl;
        std::cout<<"# i, qx  : "<<i<<", "<<qx_h<<std::endl;
        std::cout<<"#============================="<<std::endl;

        double startTotal, endTotal, timeTotal;
        timeTotal = 0.;
        startTotal = getrusage_usec();

        timeVegasCall = 0.;
        timeVegasMove = 0.;
        timeVegasFill = 0.;
        timeVegasRefine = 0.;

        double avgi = 0.;
        double sd = 0.;
        double chi2a = 0.;

        gVegas(avgi, sd, chi2a);

        endTotal = getrusage_usec();
        timeTotal = endTotal - startTotal;
        timeAllpts[i] = timeTotal;
        resultsAllpts[i] = avgi;
        errorAllpts[i] = sd;

        //-------------------------
        //  Print out information
        //-------------------------
        std::cout.clear();
        std::cout<<std::setw(10)<<std::setprecision(6)<<std::endl;
        std::cout<<"#============================="<<std::endl;
        std::cout<<"# No. of Thread Block Size  : "<<nBlockSize<<std::endl;
        std::cout<<"#============================="<<std::endl;
        std::cout<<"# No. of dimensions         : "<<ndim<<std::endl;
        std::cout<<"# No. of func calls / iter  : "<<ncall<<std::endl;
        std::cout<<"# No. of max. iterations    : "<<itmx<<std::endl;
        std::cout<<"# Desired accuracy          : "<<acc<<std::endl;
        std::cout<<"#============================="<<std::endl;
        std::cout<<std::scientific;
        std::cout<<std::left<<std::setfill(' ');
        std::cout<<"# Result                    : "
                <<std::setw(12)<<std::setprecision(5)<<avgi<<" +- "
                <<std::setw(12)<<std::setprecision(5)<<sd<<" ( "
                <<std::setw(7)<<std::setprecision(4)
                <<std::fixed<<100.*sd/avgi<<"%)"<<std::endl;
        std::cout<<std::fixed;
        std::cout<<"# Chisquare                 : "<<std::setprecision(4)
                <<chi2a<<std::endl;
        std::cout<<"#============================="<<std::endl;
        std::cout<<std::right;
        std::cout<<"# Total Execution Time(sec) : "
                <<std::setw(10)<<std::setprecision(4)<<timeTotal<<std::endl;
        std::cout<<"#============================="<<std::endl;
        std::cout<<"# Time for func calls (sec) : "
                <<std::setw(10)<<std::setprecision(4)<<timeVegasCall
                <<" ( "<<std::setw(5)<<std::setprecision(2)
                <<100.*timeVegasCall/timeTotal<<"%)"<<std::endl;
        std::cout<<"# Time for data transf (sec): "
                <<std::setw(10)<<std::setprecision(4)<<timeVegasMove
                <<" ( "<<std::setw(5)<<std::setprecision(2)
                <<100.*timeVegasMove/timeTotal<<"%)"<<std::endl;
        std::cout<<"# Time for data fill (sec)  : "
                <<std::setw(10)<<std::setprecision(4)<<timeVegasFill
                <<" ( "<<std::setw(5)<<std::setprecision(2)
                <<100.*timeVegasFill/timeTotal<<"%)"<<std::endl;
        std::cout<<"# Time for grid refine (sec): "
                <<std::setw(10)<<std::setprecision(4)<<timeVegasRefine
                <<" ( "<<std::setw(5)<<std::setprecision(2)
                <<100.*timeVegasRefine/timeTotal<<"%)"<<std::endl;
        std::cout<<"#============================="<<std::endl;

        hipDeviceReset();
  }

  std::ofstream allqxout("qxresults4_newmu.csv");

  allqxout << "i,qx,X,stddev,time" << std::endl;

  for(int i=75; i < num_qx; i++) {
        qx_h = i * PI / (a_h * num_qx);
        allqxout << i << "," << qx_h << "," << resultsAllpts[i] << "," << errorAllpts[i] << "," << timeAllpts[i] << std::endl;
  }

  allqxout.close();
  return 0;
}
