#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <omp.h>

#include "hip/hip_runtime_api.h"

#include "vegas.h"
#include "vegasconst.h"
#include "kernels.h"

#include "gvegas.h"

void gVegas(double& avgi, double& sd, double& chi2a)
{

   for (int j=0;j<ndim;j++) {
      xi[j][0] = 1.f;
   }

   // entry vegas1

   it = 0;

   // entry vegas2
   nd = nd_max;
   ng = 1;

   npg = 0;
   //std::cout<<"mds = "<<mds<<std::endl;
   if (mds!=0) {

      std::cout<<"ncall, ndim = "<<ncall<<", "<<ndim<<std::endl;
      ng = (int)pow((0.5*(double)ncall),1./(double)ndim);
      mds = 1;
      //      printf("ng = %d\n",ng);
      if (2*ng>=nd_max) {
         mds = -1;
         npg = ng/nd_max+1;
         nd = ng/npg;
         ng = npg*nd;
      }

   }
   //std::cout<<"ng = "<<ng<<std::endl;
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &ndim, sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ng),   &ng,   sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_nd),   &nd,   sizeof(int)));
   hipDeviceSynchronize(); // wait for synchronize

   nCubes = (unsigned)(pow(ng,ndim));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_nCubes), &nCubes, sizeof(nCubes)));
   hipDeviceSynchronize(); // wait for synchronize

   npg = ncall/nCubes;
   if (npg<2) npg = 2;
   calls = (double)(npg*nCubes);

   unsigned nCubeNpg = nCubes*npg;

   //   std::cout<<"nCubes= "<<nCubes<<std::endl;
   //   std::cout<<"nCubeNpg= "<<nCubeNpg<<std::endl;

   if (nprn!=0) {
      // tsi = sqrt(tsi);
      std::cout<<std::endl;
      std::cout<<" << vegas internal parameters >>"<<std::endl;
      std::cout<<"            ng: "<<std::setw(5)<<ng<<std::endl;
      std::cout<<"            nd: "<<std::setw(5)<<nd<<std::endl;
      std::cout<<"           npg: "<<std::setw(5)<<npg<<std::endl;
      std::cout<<"        nCubes: "<<std::setw(12)<<nCubes<<std::endl;
      std::cout<<"    nCubes*npg: "<<std::setw(12)<<nCubeNpg<<std::endl;
   }

   dxg = 1.f/(float)ng;
   double dnpg = (double)npg;
   double dv2g = calls*calls*pow(dxg,ndim)*pow(dxg,ndim)/(dnpg*dnpg*(dnpg-1.));
   xnd = (float)nd;
   dxg *= xnd;
   xjac = 1.f/(float)calls;
   for (int j=0;j<ndim;j++) {
      dx[j] = xu[j]-xl[j];
      xjac *= dx[j];
   }

   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_npg),  &npg,  sizeof(int)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xjac), &xjac, sizeof(float)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_dxg),  &dxg,  sizeof(float)));
   hipDeviceSynchronize(); // wait for synchronize

   ndo = 1;

   if (nd!=ndo) {

      double rc = (double)ndo/xnd;

      for (int j=0;j<ndim;j++) {

         int k = -1;
         double xn = 0.;
         double dr = 0.;
         int i = k;
         k++;
         dr += 1.;
         double xo = xn;
         xn = xi[j][k];
         //         printf("xn = %g\n",xn);
         while (i<nd-1) {

            while (dr<=rc) {
               k++;
               dr += 1.;
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn - (xn-xo)*dr;
         }

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      ndo = nd;

   }

   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xl), xl, sizeof(xl)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_dx), dx, sizeof(dx)));
   checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
   hipDeviceSynchronize(); // wait for synchronize

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << input parameters for vegas >>"<<std::endl;
      std::cout<<"     ndim ="<<std::setw(3)<<ndim
               <<"   ncall ="<<std::setw(10)<<(int)calls<<std::endl;
      std::cout<<"     it   =  0"
               <<"   itmx ="<<std::setw(5)<<itmx<<std::endl;
      std::cout<<"     acc  = "<<std::fixed
               <<std::setw(9)<<std::setprecision(3)<<acc<<std::endl;
      std::cout<<"     mds  ="<<std::setw(3)<<mds
               <<"   nd = "<<std::setw(4)<<nd<<std::endl;
      for (int j=0;j<ndim;j++) {
         std::cout<<"    (xl,xu)= ( "<<std::setw(6)<<std::fixed
                  <<xl[j]<<" , "<<xu[j]<<" )"<<std::endl;
      }

   }

   // entry vegas3

   it = 0;
   si = 0.;
   si2 = 0.;
   swgt = 0.;
   schi = 0.;
   //   int iflag;
   // main integration loop

   //   std::cout<<"nBlockSize = "<<nBlockSize<<std::endl;
   //--------------------------
   //  Set up kernel vaiables
   //--------------------------
   const int nGridSizeMax =  65535;

   dim3 ThBk(nBlockSize);

   int nGridSizeX, nGridSizeY;
   int nBlockTot = (nCubeNpg-1)/nBlockSize+1;
//   std::cout<<"nBlockTot = "<<nBlockTot<<std::endl;
   nGridSizeY = (nBlockTot-1)/nGridSizeMax+1;
   nGridSizeX = (nBlockTot-1)/nGridSizeY+1;
//   std::cout<<"nGridSize (x,y) = "<<nGridSizeX<<", "<<nGridSizeY<<std::endl;
   dim3 BkGd(nGridSizeX, nGridSizeY);

   if (nprn!=0) {
      std::cout<<std::endl;
      std::cout<<" << kernel parameters for CUDA >>"<<std::endl;
      std::cout<<"       Block size           ="<<std::setw(7)<<ThBk.x<<std::endl;
      std::cout<<"       Grid size            ="<<std::setw(7)<<BkGd.x
               <<" x "<<BkGd.y<<std::endl;
      int nThreadsTot = ThBk.x*BkGd.x*BkGd.y;
      std::cout<<"     Actual Number of calls ="<<std::setw(12)
               <<nThreadsTot<<std::endl;
      std::cout<<"   Required Number of calls ="<<std::setw(12)
               <<nCubeNpg<<" ( "<<std::setw(6)<<std::setprecision(2)
               <<100.*(double)nCubeNpg/(double)nThreadsTot<<"%)"<<std::endl;
      std::cout<<std::endl;
   }

   // allocate Fval
   int sizeFval = nCubeNpg*sizeof(float);
//   std::cout<<"sizeFval = "<<sizeFval<<std::endl;

   // CPU
   float* hFval;
   checkCudaErrors(hipHostMalloc((void**)&hFval, sizeFval));
   memset(hFval, '\0', sizeFval);

   // GPU
   float* gFval;
   checkCudaErrors(hipMalloc((void**)&gFval, sizeFval));

   // allocate IAval
   //   int sizeIAval = nCubeNpg*ndim*sizeof(unsigned short);
   int sizeIAval = nCubeNpg*ndim*sizeof(int);
//   std::cout<<"sizeIAval = "<<sizeIAval<<std::endl;

   // CPU
   //unsigned short* hIAval;
   int* hIAval;
   checkCudaErrors(hipHostMalloc((void**)&hIAval, sizeIAval));
   //unsigned short* hIAval =
   //  (unsigned short*)calloc(nCubeNpg*ndim, sizeof(unsigned short));
   memset(hIAval, '\0', sizeIAval);

   // GPU
   // unsigned short* gIAval;
   int* gIAval;
   checkCudaErrors(hipMalloc((void**)&gIAval, sizeIAval));

   double startVegasCall, endVegasCall;
   double startVegasMove, endVegasMove;
   double startVegasFill, endVegasFill;
   double startVegasRefine, endVegasRefine;

   /*Kickstart Test Functions vectors
   myVegasStartVectors<<<1, 1>>>();
   getLastCudaError("Couldn't start vectors");
   */

   do {

      it++;

//      std::cout<<"call gVegasCallFunc: it = "<<it<<std::endl;
      startVegasCall = omp_get_wtime();
      gVegasCallFunc<<<BkGd, ThBk>>>(gFval, gIAval);
      hipDeviceSynchronize(); // wait for synchronize
      endVegasCall = omp_get_wtime();
      timeVegasCall += endVegasCall-startVegasCall;

      startVegasMove = omp_get_wtime();
      checkCudaErrors(hipMemcpy(hFval, gFval,  sizeFval,
                               hipMemcpyDeviceToHost));

      checkCudaErrors(hipMemcpy(hIAval, gIAval,  sizeIAval,
                               hipMemcpyDeviceToHost));
      endVegasMove = omp_get_wtime();
      timeVegasMove += endVegasMove-startVegasMove;

// *****************

      startVegasFill = omp_get_wtime();

      ti = 0.;
      tsi = 0.;

      double d[ndim_max][nd_max];

      for (int j=0;j<ndim;++j) {
         for (int i=0;i<nd;++i) {
            d[j][i] = 0.;
         }
      }
      int maxthreads = omp_get_max_threads();
      int i, idx, ipg, iaj, idim, pieces;
      double f, f2, f2b, fb;
      unsigned ig;
    #pragma omp parallel private(iaj, idim, ig, f, f2, fb, f2b, ipg, i, idx) reduction (+:ti, tsi)
    {
      pieces = (nCubes + maxthreads - 1) / maxthreads;
      i = omp_get_thread_num();

      double d_priv[ndim_max][nd_max] = {0.};

      for (ig=i*pieces;ig<(i+1)*pieces;ig++) {
        if (ig < nCubes){
          fb = 0.;
          f2b = 0.;
         for (ipg=0;ipg<npg;ipg++) {
            idx = npg*ig+ipg;
            f = (double)hFval[idx];
            //std::cout<<"idx,f = "<<idx<<", "<<std::scientific
            //<<std::setw(10)<<std::setprecision(5)<<f<<std::endl;
            f2 = f*f;
            fb += f;
            f2b += f2;
            /*
            for (int idim=0;idim<ndim;idim++) {
               int iaj = hIAval[idim*nCubeNpg+idx];
               d[idim][iaj] += f2;
            }
            */
          }
         f2b = sqrt(f2b*npg);
         f2b = (f2b-fb)*(f2b+fb);
         ti += fb;
         tsi += f2b;
         if (mds<0) {
            for (idim=0;idim<ndim;idim++) {
               idx = npg*ig;
               iaj = hIAval[idim*nCubeNpg+idx];
               //#pragma omp atomic
               d_priv[idim][iaj] += f2b;
            }
          }
        }
      }


      if (mds>0) {
        int mdspieces = (nCubeNpg + maxthreads - 1) / maxthreads;
         //         std::cout<<"ndim = "<<ndim<<std::endl;
       for (idim=0;idim<ndim;idim++) {
          //            std::cout<<"idim = "<<idim<<std::endl;
          //#pragma omp parallel private(i, idx, iaj, f, f2) reduction (+:d)
          //{
            //i = omp_get_thread_num();
            for (idx=i*mdspieces; idx<(i+1)*mdspieces; idx++) {
               //               std::cout<<"idx = "<<idx<<std::endl;
              if (idx < nCubeNpg){
               iaj = hIAval[idim*nCubeNpg+idx];
               //               std::cout<<"iaj = "<<iaj<<std::endl;
               f = (double)hFval[idx];
               //               std::cout<<"f = "<<f<<std::endl;
               f2 = f*f;
               //#pragma omp atomic
               d_priv[idim][iaj] += f2;
               //               std::cout<<"idim, iaj, idx, f = "<<idim<<", "<<iaj
               //                        <<", "<<idx<<", "<<f<<std::endl;
              }
            }
          //}
        }
      }

      #pragma omp critical
      {
        for (int h = 0; h < ndim; h++){
          for (int j = 0; j < nd; j++){
            d[h][j] += d_priv[h][j];
          }
        }
      }

    }

      endVegasFill = omp_get_wtime();
      timeVegasFill += endVegasFill-startVegasFill;

      tsi *= dv2g;
      double ti2 = ti*ti;
      double wgt = ti2/tsi;
      si += ti*wgt;
      si2 += ti2;
      swgt += wgt;
      schi += ti2*wgt;
      avgi = si/swgt;
      sd = swgt*it/si2;
      chi2a = 0.;
      if (it>1) chi2a = sd*(schi/swgt-avgi*avgi)/((double)it-1.);
      sd = sqrt(1./sd);

      if (nprn!=0) {
         tsi = sqrt(tsi);
         std::cout<<std::endl;
         std::cout<<" << integration by vegas >>"<<std::endl;
         std::cout<<"     iteration no. "<<std::setw(4)<<it
                  <<"   integral=  "<<ti<<std::endl;
         std::cout<<"                          std dev  = "<<tsi<<std::endl;
         std::cout<<"     accumulated results: integral = "<<avgi<<std::endl;
         std::cout<<"                          std dev  = "<<sd<<std::endl;
	 if (it > 1) {
            std::cout<<"                          chi**2 per it'n = "
                     <<std::setw(10)<<std::setprecision(6)<<chi2a<<std::endl;
         }
         if (nprn<0) {
            for (int j=0;j<ndim;j++) {
               std::cout<<"   == data for axis "
                        <<std::setw(2)<<j<<" --"<<std::endl;
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce";
               std::cout<<"    x    delt i   convce"<<std::endl;
               /*
               for (int i=0;i<nd;i+=3) {
                  std::cout<<std::setw(6)<<std::setprecision(2)<<std::setfill(' ')
                           <<xi[j][i]<<" "<<di[j][i]<<" "<<d[j][i];
                  std::cout<<std::setw(6)<<std::setprecision(2)
                           <<xi[j][i+1]<<" "<<di[j][i+1]<<" "<<d[j][i+1];
                  std::cout<<std::setw(6)<<std::setprecision(2)
                           <<xi[j][i+2]<<" "<<di[j][i+2]<<" "<<d[j][i+2]
                           <<std::endl;
                           }
               */
            }
         }
      }

      // refine grid

      startVegasRefine = omp_get_wtime();

      /*
      for (int ii=0;ii<ndim;ii++) {
         for (int jj=0;jj<nd;jj++) {
            std::cout<<"d["<<ii<<"]["<<jj<<"] = "<<std::scientific
                     <<d[ii][jj]<<std::endl;
         }
      }
      */

      double r[nd_max];
      double dt[ndim_max];
      for (int j=0;j<ndim;j++) {
         double xo = d[j][0];
         double xn = d[j][1];
         d[j][0] = 0.5*(xo+xn);
         dt[j] = d[j][0];
         for (int i=1;i<nd-1;i++) {
            d[j][i] = xo+xn;
            xo = xn;
            xn = d[j][i+1];
            d[j][i] = (d[j][i]+xn)/3.;
            dt[j] += d[j][i];
         }
         d[j][nd-1] = 0.5*(xn+xo);
         dt[j] += d[j][nd-1];
      }

      for (int j=0;j<ndim;j++) {
         double rc = 0.;
         for (int i=0;i<nd;i++) {
            r[i] = 0.;
            if (d[j][i]>0.) {
               double xo = dt[j]/d[j][i];
               if (!isinf(xo))
                  r[i] = pow(((xo-1.)/xo/log(xo)),alph);
            }
            rc += r[i];
         }
         rc /= xnd;
         int k = -1;
         double xn = 0.;
         double dr = xn;
         int i = k;
         k++;
         dr += r[k];
         double xo = xn;
         xn = xi[j][k];

         do {

            while (dr<=rc) {
               k++;
               dr += r[k];
               xo = xn;
               xn = xi[j][k];
            }
            i++;
            dr -= rc;
            xin[i] = xn-(xn-xo)*dr/r[k];

         } while (i<nd-2);

         for (int i=0;i<nd-1;i++) {
            xi[j][i] = (float)xin[i];
         }
         xi[j][nd-1] = 1.f;

      }
      checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_xi), xi, sizeof(xi)));
      hipDeviceSynchronize(); // wait for synchronize

      endVegasRefine = omp_get_wtime();
      timeVegasRefine += endVegasRefine-startVegasRefine;

//      std::cout<<"The end of main loop: it, sd/avgi = "<<it<<", "
//               <<sd/fabs(avgi)<<std::endl;

   } while (it<itmx && acc*fabs(avgi)<sd);


   checkCudaErrors(hipHostFree(hFval));
   checkCudaErrors(hipFree(gFval));

   checkCudaErrors(hipHostFree(hIAval));
//   free(hIAval);
   checkCudaErrors(hipFree(gIAval));

   //   std::cout<<"ng = "<<ng<<std::endl;
}
