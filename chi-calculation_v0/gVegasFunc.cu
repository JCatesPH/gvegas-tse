#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

/*-------- constants for chi ---------*/
#define mu      0.1f
#define hOmg    0.3f
#define a       4.f
#define A       4.f
#define rati    0.3
#define eE0     rati * (hOmg * hOmg) / (2 * sqrt(A * mu))
#define Gamm    0.003
#define KT      1e-6
#define shift   A * (eE0 / hOmg) * (eE0 / hOmg)
#define Gammsq  Gamm * Gamm
#define N       3

/*-------- helpful macros ---------*/
#define SQ(x)  (x * x) // Squares the argument
#define CB(x)  (x * x * x) // Cubes the argument

__device__
float chi(float* rx, float wgt)
{
   float dds = 0.f;
   // ds = 0  // UNUSED
   float ek;
   float ekq;
   float xk;
   float xkq;

   // ek = A * (sqrt((rx[0]) ** 2 + (rx[1]) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ek = A * hypotf(rx[0], rx[1]) * hypotf(rx[0], rx[1]) + A * SQ(eE0 / hOmg);

   // ekq = A * (sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ekq = A * hypotf(rx[0] + rx[2], rx[1]) * hypotf(rx[0] + rx[2], rx[1]) + A * SQ(eE0 / hOmg);

   // xk = 2 * A * eE0 * sqrt((rx[0]) ** 2 + (rx[1]) ** 2) / hOmg ** 2
   xk = 2 * A * eE0 * hypotf(rx[0], rx[1]) / SQ(hOmg);

   // xkq = 2 * A * eE0 * sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2) / hOmg ** 2
   xkq = 2 * A * eE0 * hypotf(rx[0] + rx[2], rx[1]) / SQ(hOmg);

   // singmatrix = numba.cuda.shared.array((10,N),dtype=numba.types.complex128)
   hipFloatComplex* sing;
   checkCudaErrors(hipMalloc((void**)&sing, N*sizeof(hipFloatComplex)));

   n = 0
   for (int j=-(N - 1)/2; i < ((N-1)/2+1)); i++) {
      sing[0 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ek - hOmg / 2 + hOmg * i), 0);
      sing[1 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ekq - hOmg / 2 + hOmg * i), 0);
      sing[2 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ek + hOmg / 2 + hOmg * i), 0);
      sing[3 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ekq + hOmg / 2 + hOmg * i), 0);

      sing[8 + n * 6] = make_hipFloatComplex(step(mu - hOmg / 2 - hOmg * i, 0.f), 0); 
      sing[9 + n * 6] = make_hipFloatComplex(step(mu + hOmg / 2 - hOmg * i, 0.f), 0); 

      //------//

      sing[4 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek - hOmg / 2 + hOmg * i)));  
      sing[5 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq - hOmg / 2 + hOmg * i))); 
      sing[6 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek + hOmg / 2 + hOmg * i)));  
      sing[7 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq + hOmg / 2 + hOmg * i))); 

      n = n + 1
   }

   hipFloatComplex* dbl;
   checkCudaErrors(hipMalloc((void**)&dbl, 2*(2*N-1)*sizeof(hipFloatComplex)));

   n = 0
   for (int i=-(N - 1); i < N; i++){
       dbl[0 + n * 5] = make_hipFloatComplex(2 * atan2f(Gamm, (ek - mu + hOmg * i)), 0);
       dbl[1 + n * 5] = make_hipFloatComplex(2 * atan2f(Gamm, (ekq - mu + hOmg * i)), 0);

       dbl[4 + n * 5] = make_hipFloatComplex(jnf(i, xk), 0); // Bessel function of order i
       dbl[5 + n * 5] = make_hipFloatComplex(jnf(i, xkq), 0);

       dbl[6 + n * 5] = make_hipFloatComplex(ek - ekq + hOmg * i, 0);

       //------//

       dbl[2 + n * 2] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek - mu + hOmg * i)));
       dbl[3 + n * 2] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq - mu + hOmg * i)));

       //------//

       dbl[7 + n * 2] = make_hipFloatComplex(ek - ekq + hOmg * i, 2 * Gamm);
       dbl[8 + n * 2] = make_hipFloatComplex(hOmg * i, 2 * Gamm);

       n = n + 1
    }

    I2 = make_hipFloatComplex(0, -2);

   for (int n=0; n<N; n++){
       for (int alpha=0; alpha<N; alpha++){
           for (int beta=0; beta<N; beta++){
               for (int gamma=0; gamma<N; gamma++){
                   for (int s=0; s<N; s++){
                       for (int l=0; l<N; l++){
                            // p1p = dbl[6+(beta - gamma + N - 1)*9] * (sing[0+alpha*10] - dbl[0+(s+alpha)*9] - sing[4+alpha*10] + dbl[2+(s+alpha)*9])
                            // p2p = dbl[7+(alpha-gamma+N-1)*9] * (sing[0+beta*10] - dbl[0+(s+beta)*9] + sing[4+beta*10] - dbl[2+(s+beta)*9])
                            // p3p = dbl[8+(alpha-beta+N-1)*9] * (-sing[1+gamma*10] + dbl[1+(s+gamma)*9] - sing[5+gamma*10] + dbl[3+(s+gamma)*9])

                            // p1m = dbl[6 + (beta - gamma + N - 1) * 9] * (sing[2+alpha*10] - dbl[(s+alpha)*9] - sing[6+alpha*10] + dbl[2+(s+alpha)*9])
                            // p2m = dbl[7+(alpha-gamma+N-1)*9] * ( sing[2+beta*10] - dbl[0+(s+beta)*9] + sing[6+beta*10] - dbl[2+(s+beta)*9])
                            // p3m = dbl[8+(alpha-beta+N-1)*9] * (-sing[3+gamma*10] + dbl[1+(s+gamma)*9] - sing[7+gamma*10] + dbl[3+(s+gamma)*9])

                            // d1 = hipCmulf(2I, hipCmulf(dbl[6 + (beta - gamma + N - 1) * 9], hipCmulf(dbl[7+(alpha-gamma+N-1)*9], dbl[8+(alpha-beta+N-1)*9])))

                            omint1p = hipCmulf(
                                sing[8+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[6+(beta - gamma + N - 1)*9], 
                                                hipCsubf(
                                                    hipCsubf(sing[alpha*10], dbl[(s+alpha)*9]), 
                                                    hipCaddf(sing[4+alpha*10], dbl[2+(s+alpha)*9])
                                                )
                                            ), 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                hipCaddf(
                                                    hipCsubf(sing[beta*10], dbl[(s+beta)*9]), 
                                                    hipCsubf(sing[4+beta*10], dbl[2+(s+beta)*9])
                                                )
                                            )
                                        ), 
                                        hipCmulf(
                                            dbl[8+(alpha-beta+N-1)*9], 
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[1+gamma*10]), 
                                                hipCaddf(sing[5+gamma*10], dbl[3+(s+gamma)*9])
                                            )
                                        ),
                                    ),  
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6 + (beta - gamma + N - 1) * 9], 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                dbl[8+(alpha-beta+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            )

                            omint1m = hipCmulf(
                                sing[9+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[6+(beta-gamma+N-1) * 9], 
                                                hipCsubf(
                                                    hipCsubf(sing[2+alpha*10], dbl[(s+alpha)*9]), 
                                                    hipCaddf(sing[6+alpha*10], dbl[2+(s+alpha)*9])
                                                )
                                            ),  
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                hipCsubf(
                                                    hipCsubf(sing[2+beta*10], dbl[(s+beta)*9]), 
                                                    hipCaddf(sing[6+beta*10], dbl[2+(s+beta)*9])
                                                )
                                            )
                                        ), 
                                        hipCmulf(
                                            dbl[8+(alpha-beta+N-1)*9], 
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[3+gamma*10]), 
                                                hipCaddf(sing[7+gamma*10] + dbl[3+(s+gamma)*9])
                                            )
                                        )
                                    ),  
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6 + (beta - gamma + N - 1) * 9], 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                dbl[8+(alpha-beta+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            )

                            bess1 = hipCmulf(
                                dbl[5+(gamma-n+N-1)*9], 
                                hipCmulf(
                                    dbl[5+(gamma-l+N-1)*9], 
                                    hipCmulf(
                                        dbl[4+(beta-l+N-1)*9], 
                                        hipCmulf(
                                            dbl[4+(beta-s+N-1)*9], 
                                            hipCmulf(
                                                dbl[4+(alpha-s+N-1)*9], 
                                                dbl[4+(alpha-n+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            )
                            grgl = hipCmulf(bess1, hipCsubf(omint1p, omint1m))

                            pp1p = hipCmulf(
                                dbl[6+(alpha-beta+N-1)*9], 
                                hipCsubf(
                                    hipCsubf(dbl[1+(s+gamma)*9], sing[1+gamma*10]), 
                                    hipCaddf(sing[5+gamma*10], dbl[3+(s+gamma)*9])
                                )
                            )

                            pp2p = hipCmulf(
                                dbl[7+(alpha-beta+N-1)*9],
                                hipCaddf(
                                    hipCsubf(dbl[1+(s+beta)*9], sing[1+beta*10]), 
                                    hipCsubf(sing[5+beta*10], dbl[3+(s+beta)*9])
                                )
                            )

                            pp3p = hipCmulf(
                                dbl[8+(beta-gamma+N-1)*9],
                                hipCsubf(
                                    hipCsubf(sing[alpha*10], dbl[0 + (s+alpha) * 9]), 
                                    hipCaddf(sing[4+alpha*10], dbl[2 + (s+alpha) * 9])
                                )
                            )

                            pp1m = hipCmulf(
                                dbl[6+(alpha-beta+N-1)*9],
                                hipCsubf(
                                    hipCsubf(dbl[1+(s+gamma)*9], sing[3+gamma*10]),
                                    hipCaddf(sing[7+gamma*10], dbl[3+(s+gamma)*9])
                                )
                            )

                            pp2m = hipCmulf(
                                dbl[7+(alpha-gamma+N-1)*9], 
                                hipCaddf(
                                    hipCsubf(dbl[1+(s+beta)*9], sing[3+beta*10]), 
                                    hipCsubf(sing[7+beta*10], dbl[3+(s+beta)*9])
                                )
                            )

                            pp3m = hipCmulf(
                                dbl[8+(beta-gamma+N-1)*9],
                                hipCsubf(
                                    hipCsubf(sing[2+alpha*10], dbl[0 + (s+alpha) * 9]),
                                    hipCaddf(sing[6+alpha*10], dbl[2 + (s+alpha) * 9])
                                )
                            )

                            d2 = hipCmulf(
                                I2, 
                                hipCmulf(
                                    dbl[6+(alpha-beta+N-1)*9],
                                    hipCmulf(
                                        dbl[7+(alpha-gamma+N-1)*9],
                                        dbl[8+(beta-gamma+N-1)*9]
                                    )
                                )
                            )
                            omint2p = hipCmulf(
                                sing[8+s*10],
                                hipCdivf(
                                    hipCaddf(
                                        pp1p, 
                                        hipCaddf(
                                            pp2p, 
                                            pp3p
                                        ),
                                    ) 
                                    d2
                                )
                            )

                            omint2m = hipCmulf(
                                sing[9+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        pp1m, 
                                        hipCaddf(
                                            pp2m, 
                                            pp3m
                                        ),
                                    ) 
                                    d2
                                )
                            )

                            bess2 = hipCmulf(
                                dbl[5+(gamma-n+N-1)*9],
                                hipCmulf(
                                    dbl[5+(gamma-s+N-1)*9], 
                                    hipCmulf(
                                        dbl[5+(beta-s+N-1)*9],
                                        hipCmulf(
                                            dbl[5+(beta-l+N-1)*9],
                                            hipCmulf(
                                                dbl[4+(alpha-l+N-1)*9],
                                                dbl[4+(alpha-n+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            )
                            glga = hipCmulf(
                                bess2, 
                                hipCsubf(omint2p, omint2m)
                            )

                            dds = hipCaddf(
                                dds, 
                                // Gamm * (grgl + glga)
                                hipCaddf(
                                    make_hipFloatComplex(Gamm*hipCrealf(grgl), Gamm*hipCimagf(grgl)),
                                    make_hipFloatComplex(Gamm*hipCrealf(glga), Gamm*hipCimagf(glga))
                                )
                        }
                    }
                }
            }
        }
    }
   return -8 * hipCrealf(dds) / CB(HIP_PI_F);
}
