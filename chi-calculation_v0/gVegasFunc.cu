#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

/*-------- constants for chi ---------*/
#define mu      0.1f
#define hOmg    0.3f
#define a       4.f
#define A       4.f
#define rati    0.3
#define eE0     rati * (hOmg * hOmg) / (2 * sqrt(A * mu))
#define Gamm    0.003
#define KT      1e-6
#define shift   A * (eE0 / hOmg) * (eE0 / hOmg)
#define Gammsq  Gamm * Gamm
#define N       3

/*-------- helpful macros ---------*/
#define SQ(x)  (x * x) // Squares the argument
#define CB(x)  (x * x * x) // Cubes the argument

__device__
float chi(float* rx, float wgt)
{
   float dds = 0.f;
   // ds = 0  // UNUSED
   float ek;
   float ekq;
   float xk;
   float xkq;

   // ek = A * (sqrt((rx[0]) ** 2 + (rx[1]) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ek = A * hypotf(rx[0], rx[1]) * hypotf(rx[0], rx[1]) + A * SQ(eE0 / hOmg);

   // ekq = A * (sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ekq = A * hypotf(rx[0] + rx[2], rx[1]) * hypotf(rx[0] + rx[2], rx[1]) + A * SQ(eE0 / hOmg);

   // xk = 2 * A * eE0 * sqrt((rx[0]) ** 2 + (rx[1]) ** 2) / hOmg ** 2
   xk = 2 * A * eE0 * hypotf(rx[0], rx[1]) / SQ(hOmg);

   // xkq = 2 * A * eE0 * sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2) / hOmg ** 2
   xkq = 2 * A * eE0 * hypotf(rx[0] + rx[2], rx[1]) / SQ(hOmg);

   // singmatrix = numba.cuda.shared.array((10,N),dtype=numba.types.complex128)
   int sizesing = 10 * N * sizeof(float);
   float* singmatrix;
   checkCudaErrors(hipMalloc((void**)&singmatrix, sizesing));

   n = 0
   for (int j=-(N - 1)/2; i < ((N-1)/2+1)); i++) {
      singmatrix[0 + n * 10] = 2 * atan2f(Gamm, ek - hOmg / 2 + hOmg * i);
      singmatrix[1 + n * 10] = 2 * atan2f(Gamm, ekq - hOmg / 2 + hOmg * i);
      singmatrix[2 + n * 10] = 2 * atan2f(Gamm, ek + hOmg / 2 + hOmg * i);
      singmatrix[3 + n * 10] = 2 * atan2f(Gamm, ekq + hOmg / 2 + hOmg * i);

//========================================================================================================//
// HERE DOWN NEEDS MODIFICATION
      singmatrix[4 + n * 10] = complex(0, 1) * logf(Gammsq + SQ(ek - hOmg / 2 + hOmg * i));
      singmatrix[5 + n * 10] = complex(0, 1) * logf(Gammsq + SQ(ekq - hOmg / 2 + hOmg * i));
      singmatrix[6 + n * 10] = complex(0, 1) * logf(Gammsq + SQ(ek + hOmg / 2 + hOmg * i));
      singmatrix[7 + n * 10] = complex(0, 1) * logf(Gammsq + SQ(ekq + hOmg / 2 + hOmg * i));

      singmatrix[8 + n * 10] = cudahelpers.my_heaviside(mu - hOmg / 2 - hOmg * i);
      singmatrix[9 + n * 10] = cudahelpers.my_heaviside(mu + hOmg / 2 - hOmg * i);
      n = n + 1
   }

   size_dbl = 5
   dblmatrix = numba.cuda.shared.array((9,size_dbl),dtype=numba.types.complex128)

   n = 0
   for i in range(-(N - 1), N, 1):
       xi = hOmg * i
       zeta = ek - mu + xi
       eta = ekq - mu + xi

       zetasq = zeta ** 2
       etasq = eta ** 2

       dblmatrix[0,n] = 2 * atan2f(Gamm, zeta)
       dblmatrix[1,n] = 2 * atan2f(Gamm, eta)

       logged1 = logf(Gammsq + zetasq)
       logged2 = logf(Gammsq + etasq)

       dblmatrix[2,n] = complex(0, logged1)
       dblmatrix[3,n] = complex(0, logged2)

       dblmatrix[4,n] = cudahelpers.besselj(i, xk)
       dblmatrix[5,n] = cudahelpers.besselj(i, xkq)

       fac1i = ek - ekq + xi
       fac2i = complex(fac1i, 2 * Gamm)
       dblmatrix[6,n] = fac1i
       dblmatrix[7,n] = fac2i
       dblmatrix[8,n] = fac2i - ek + ekq
       n = n + 1

   #numba.cuda.syncthreads()

   for n in range(0, N):
       for alpha in range(0, N):
           for beta in range(0, N):
               for gamma in range(0, N):
                   for s in range(0, N):
                       for l in range(0, N):
                           p1p = dblmatrix[6,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblmatrix[0,s + alpha] - singmatrix[4,alpha] + dblmatrix[2,s + alpha])
                           p2p = dblmatrix[7,alpha - gamma + N - 1] * (singmatrix[0,beta] - dblmatrix[0,s + beta] + singmatrix[4,beta] - dblmatrix[2,s + beta])
                           p3p = dblmatrix[8,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblmatrix[1,s + gamma] - singmatrix[5,gamma] + dblmatrix[3,s + gamma])

                           p1m = dblmatrix[6,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblmatrix[0,s + alpha] - singmatrix[6,alpha] + dblmatrix[2,s + alpha])

                           p2m = dblmatrix[7,alpha - gamma + N - 1] * ( singmatrix[2,beta] - dblmatrix[0,s + beta] + singmatrix[6,beta] - dblmatrix[2,s + beta])

                           p3m = dblmatrix[8,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblmatrix[1,s + gamma] - singmatrix[7,gamma] + dblmatrix[3,s + gamma])

                           d1 = -2 * complex(0, 1) * dblmatrix[6,beta - gamma + N - 1] * dblmatrix[7,alpha - gamma + N - 1] * dblmatrix[8,alpha - beta + N - 1]

                           omint1p = singmatrix[8,s] * ((p1p + p2p + p3p) / d1)

                           omint1m = singmatrix[9,s] * ((p1m + p2m + p3m) / d1)

                           bess1 = dblmatrix[5,gamma - n + N - 1] * dblmatrix[5,gamma - l + N - 1] * dblmatrix[4,beta - l + N - 1] * dblmatrix[4,beta - s + N - 1] * dblmatrix[4,alpha - s + N - 1] * dblmatrix[4,alpha - n + N - 1]

                           grgl = bess1 * (omint1p - omint1m)

                           pp1p = dblmatrix[6,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblmatrix[1,s + gamma] - singmatrix[5,gamma] + dblmatrix[3,s + gamma])

                           pp2p = dblmatrix[7,alpha - gamma + N - 1] * (-singmatrix[1,beta] + dblmatrix[1,s + beta] + singmatrix[5,beta] - dblmatrix[3,s + beta])

                           pp3p = dblmatrix[8,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblmatrix[0,s + alpha] - singmatrix[4,alpha] + dblmatrix[2,s + alpha])

                           pp1m = dblmatrix[6,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblmatrix[1,s + gamma] - singmatrix[7,gamma] + dblmatrix[3,s + gamma])

                           pp2m = dblmatrix[7,alpha - gamma + N - 1] * (-singmatrix[3,beta] + dblmatrix[1,s + beta] + singmatrix[7,beta] - dblmatrix[3,s + beta])

                           pp3m = dblmatrix[8,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblmatrix[0,s + alpha] - singmatrix[6,alpha] + dblmatrix[2,s + alpha])

                           d2 = -2 * complex(0, 1) * dblmatrix[6,alpha - beta + N - 1] * dblmatrix[7,alpha - gamma + N - 1] * dblmatrix[8,beta - gamma + N - 1]

                           omint2p = singmatrix[8,s] * ((pp1p + pp2p + pp3p) / d2)

                           omint2m = singmatrix[9,s] * ((pp1m + pp2m + pp3m) / d2)

                           bess2 = dblmatrix[5,gamma - n + N - 1] * dblmatrix[5,gamma - s + N - 1] * dblmatrix[5,beta - s + N - 1] * dblmatrix[5,beta - l + N - 1] * dblmatrix[4,alpha - l + N - 1] * dblmatrix[4,alpha - n + N - 1]

                           glga = bess2 * (omint2p - omint2m)

                           dds = dds + Gamm * (grgl + glga)
   return -8 * dds.real / CB(HIP_PI_F);
}
