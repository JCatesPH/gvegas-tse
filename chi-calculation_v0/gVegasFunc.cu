#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

/*-------- constants for chi ---------*/
#define mu      0.1f
#define hOmg    0.3f
#define a       3.6f
#define A       4.f
#define rati    0.1
#define eE0     rati * (hOmg * hOmg) / (2 * sqrt(A * mu))
#define Gamm    0.003
#define KT      1e-6
#define shift   A * (eE0 / hOmg) * (eE0 / hOmg)
#define Gammsq  Gamm * Gamm
#define N       3

/*-------- helpful macros ---------*/
#define SQ(x)  (x * x) // Squares the argument
#define CB(x)  (x * x * x) // Cubes the argument

__device__
float heaviside(float x, float z)
{
    if (x < z)
    {
        return 0.f;
    }
    else
    {
        return 1.f;
    }
    
}

__device__
float chi(float* rx, float wgt)
{
   // ds = 0  // UNUSED
   float ek;
   float ekq;
   float xk;
   float xkq;

   // ek = A * (sqrt((rx[0]) ** 2 + (rx[1]) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ek = A * hypotf(rx[0], rx[1]) * hypotf(rx[0], rx[1]) + A * SQ(eE0 / hOmg);

   // ekq = A * (sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ekq = A * hypotf(rx[0] + rx[2], rx[1]) * hypotf(rx[0] + rx[2], rx[1]) + A * SQ(eE0 / hOmg);

   // xk = 2 * A * eE0 * sqrt((rx[0]) ** 2 + (rx[1]) ** 2) / hOmg ** 2
   xk = 2 * A * eE0 * hypotf(rx[0], rx[1]) / SQ(hOmg);

   // xkq = 2 * A * eE0 * sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2) / hOmg ** 2
   xkq = 2 * A * eE0 * hypotf(rx[0] + rx[2], rx[1]) / SQ(hOmg);

   // singmatrix = numba.cuda.shared.array((10,N),dtype=numba.types.complex128)
   hipFloatComplex* sing;
   hipMalloc((void**)&sing, N*sizeof(hipFloatComplex));

   int n = 0;
   for (int i=-(N-1)/2; i<((N-1)/2+1); i++) {
    sing[0 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ek - hOmg / 2 + hOmg * i), 0);
    sing[1 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ekq - hOmg / 2 + hOmg * i), 0);
    sing[2 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ek + hOmg / 2 + hOmg * i), 0);
    sing[3 + n * 6] = make_hipFloatComplex(2 * atan2f(Gamm, ekq + hOmg / 2 + hOmg * i), 0);

    sing[8 + n * 6] = make_hipFloatComplex(heaviside(mu - hOmg / 2 - hOmg * i, 0.f), 0); 
    sing[9 + n * 6] = make_hipFloatComplex(heaviside(mu + hOmg / 2 - hOmg * i, 0.f), 0); 

    sing[4 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek - hOmg / 2 + hOmg * i)));  
    sing[5 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq - hOmg / 2 + hOmg * i))); 
    sing[6 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek + hOmg / 2 + hOmg * i)));  
    sing[7 + n * 10] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq + hOmg / 2 + hOmg * i))); 

    n = n + 1;
   }

   hipFloatComplex* dbl;
   hipMalloc((void**)&dbl, 9*(2*N-1)*sizeof(hipFloatComplex));

   n = 0;
   for (int i=-(N-1); i < N; i++)
   {
       dbl[0 + n * 5] = make_hipFloatComplex(2 * atan2f(Gamm, (ek - mu + hOmg * i)), 0);
       dbl[1 + n * 5] = make_hipFloatComplex(2 * atan2f(Gamm, (ekq - mu + hOmg * i)), 0);

       dbl[4 + n * 5] = make_hipFloatComplex(jnf(i, xk), 0); // Bessel function of order i
       dbl[5 + n * 5] = make_hipFloatComplex(jnf(i, xkq), 0);

       dbl[6 + n * 5] = make_hipFloatComplex(ek - ekq + hOmg * i, 0);

       dbl[2 + n * 2] = make_hipFloatComplex(0, logf(Gammsq + SQ(ek - mu + hOmg * i)));
       dbl[3 + n * 2] = make_hipFloatComplex(0, logf(Gammsq + SQ(ekq - mu + hOmg * i)));

       dbl[7 + n * 2] = make_hipFloatComplex(ek - ekq + hOmg * i, 2 * Gamm);
       dbl[8 + n * 2] = make_hipFloatComplex(hOmg * i, 2 * Gamm);

       n = n + 1;
    }

    hipFloatComplex I2 = make_hipFloatComplex(0, -2);

    hipFloatComplex omint1p;
    hipFloatComplex omint1m;
    hipFloatComplex bess1;
    hipFloatComplex omint2p;
    hipFloatComplex omint2m;
    hipFloatComplex bess2;
    hipFloatComplex dds;


   for (int n=0; n<N; n++){
       for (int alpha=0; alpha<N; alpha++){
           for (int beta=0; beta<N; beta++){
               for (int gamma=0; gamma<N; gamma++){
                   for (int s=0; s<N; s++){
                       for (int l=0; l<N; l++){
                            omint1p = hipCmulf(
                                sing[8+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[6+(beta - gamma + N - 1)*9], 
                                                hipCsubf(
                                                    hipCsubf(sing[alpha*10], dbl[(s+alpha)*9]), 
                                                    hipCaddf(sing[4+alpha*10], dbl[2+(s+alpha)*9])
                                                )
                                            ), 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                hipCaddf(
                                                    hipCsubf(sing[beta*10], dbl[(s+beta)*9]), 
                                                    hipCsubf(sing[4+beta*10], dbl[2+(s+beta)*9])
                                                )
                                            )
                                        ), 
                                        hipCmulf(
                                            dbl[8+(alpha-beta+N-1)*9], 
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[1+gamma*10]), 
                                                hipCaddf(sing[5+gamma*10], dbl[3+(s+gamma)*9])
                                            )
                                        )
                                    ),  
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6 + (beta - gamma + N - 1) * 9], 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                dbl[8+(alpha-beta+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            omint1m = hipCmulf(
                                sing[9+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[6+(beta-gamma+N-1) * 9], 
                                                hipCsubf(
                                                    hipCsubf(sing[2+alpha*10], dbl[(s+alpha)*9]), 
                                                    hipCaddf(sing[6+alpha*10], dbl[2+(s+alpha)*9])
                                                )
                                            ),  
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                hipCsubf(
                                                    hipCsubf(sing[2+beta*10], dbl[(s+beta)*9]), 
                                                    hipCaddf(sing[6+beta*10], dbl[2+(s+beta)*9])
                                                )
                                            )
                                        ), 
                                        hipCmulf(
                                            dbl[8+(alpha-beta+N-1)*9], 
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[3+gamma*10]), 
                                                hipCaddf(sing[7+gamma*10], dbl[3+(s+gamma)*9])
                                            )
                                        )
                                    ),  
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6 + (beta - gamma + N - 1) * 9], 
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                dbl[8+(alpha-beta+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            bess1 = hipCmulf(
                                dbl[5+(gamma-n+N-1)*9], 
                                hipCmulf(
                                    dbl[5+(gamma-l+N-1)*9], 
                                    hipCmulf(
                                        dbl[4+(beta-l+N-1)*9], 
                                        hipCmulf(
                                            dbl[4+(beta-s+N-1)*9], 
                                            hipCmulf(
                                                dbl[4+(alpha-s+N-1)*9], 
                                                dbl[4+(alpha-n+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            omint2p = hipCmulf(
                                sing[8+s*10],
                                hipCdivf(
                                    hipCaddf(
                                        hipCmulf(
                                            dbl[6+(alpha-beta+N-1)*9], 
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[1+gamma*10]), 
                                                hipCaddf(sing[5+gamma*10], dbl[3+(s+gamma)*9])
                                            )
                                        ), 
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[7+(alpha-beta+N-1)*9],
                                                hipCaddf(
                                                    hipCsubf(dbl[1+(s+beta)*9], sing[1+beta*10]), 
                                                    hipCsubf(sing[5+beta*10], dbl[3+(s+beta)*9])
                                                )
                                            ), 
                                            hipCmulf(
                                                dbl[8+(beta-gamma+N-1)*9],
                                                hipCsubf(
                                                    hipCsubf(sing[alpha*10], dbl[0 + (s+alpha) * 9]), 
                                                    hipCaddf(sing[4+alpha*10], dbl[2 + (s+alpha) * 9])
                                                )
                                            )
                                        )
                                    ), 
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6+(alpha-beta+N-1)*9],
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9],
                                                dbl[8+(beta-gamma+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            omint2m = hipCmulf(
                                sing[9+s*10], 
                                hipCdivf(
                                    hipCaddf(
                                        hipCmulf(
                                            dbl[6+(alpha-beta+N-1)*9],
                                            hipCsubf(
                                                hipCsubf(dbl[1+(s+gamma)*9], sing[3+gamma*10]),
                                                hipCaddf(sing[7+gamma*10], dbl[3+(s+gamma)*9])
                                            )
                                        ), 
                                        hipCaddf(
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9], 
                                                hipCaddf(
                                                    hipCsubf(dbl[1+(s+beta)*9], sing[3+beta*10]), 
                                                    hipCsubf(sing[7+beta*10], dbl[3+(s+beta)*9])
                                                )
                                            ), 
                                            hipCmulf(
                                                dbl[8+(beta-gamma+N-1)*9],
                                                hipCsubf(
                                                    hipCsubf(sing[2+alpha*10], dbl[0 + (s+alpha) * 9]),
                                                    hipCaddf(sing[6+alpha*10], dbl[2 + (s+alpha) * 9])
                                                )
                                            )
                                        )
                                    ),
                                    hipCmulf(
                                        I2, 
                                        hipCmulf(
                                            dbl[6+(alpha-beta+N-1)*9],
                                            hipCmulf(
                                                dbl[7+(alpha-gamma+N-1)*9],
                                                dbl[8+(beta-gamma+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            bess2 = hipCmulf(
                                dbl[5+(gamma-n+N-1)*9],
                                hipCmulf(
                                    dbl[5+(gamma-s+N-1)*9], 
                                    hipCmulf(
                                        dbl[5+(beta-s+N-1)*9],
                                        hipCmulf(
                                            dbl[5+(beta-l+N-1)*9],
                                            hipCmulf(
                                                dbl[4+(alpha-l+N-1)*9],
                                                dbl[4+(alpha-n+N-1)*9]
                                            )
                                        )
                                    )
                                )
                            );

                            dds = hipCaddf(
                                dds, 
                                // Gamm * (grgl + glga)
                                hipCaddf(
                                    make_hipFloatComplex(
                                        Gamm*hipCrealf(hipCmulf(
                                            bess1, 
                                            hipCsubf(omint1p, omint1m)
                                            )
                                        ), 
                                        Gamm*hipCimagf(hipCmulf(
                                            bess1, 
                                            hipCsubf(omint1p, omint1m))
                                            )
                                        ),
                                    make_hipFloatComplex(
                                        Gamm*hipCrealf(hipCmulf(
                                            bess2, 
                                            hipCsubf(omint2p, omint2m)
                                            )
                                        ), 
                                        Gamm*hipCimagf(hipCmulf(
                                            bess2, 
                                            hipCsubf(omint2p, omint2m))
                                        )
                                    )
                                )
                            );
                        }
                    }
                }
            }
        }
    }
   return -8 * hipCrealf(dds) / CB(HIP_PI_F);
}
