#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

/*-------- constants for chi ---------*/
#define mu      0.1f
#define hOmg    0.3f
#define a       4.f
#define A       4.f
#define rati    0.3
#define eE0     rati * (hOmg * hOmg) / (2 * sqrt(A * mu))
#define Gamm    0.003
#define KT      1e-6
#define shift   A * (eE0 / hOmg) * (eE0 / hOmg)
#define Gammsq  Gamm * Gamm
#define N       3

/*-------- helpful macros ---------*/
#define SQ(x)  (x * x) // Squares the argument
#define CB(x)  (x * x * x) // Cubes the argument

__device__
float chi(float* rx, float wgt)
{
   float dds = 0.f;
   // ds = 0  // UNUSED
   float ek;
   float ekq;
   float xk;
   float xkq;

   // ek = A * (sqrt((rx[0]) ** 2 + (rx[1]) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ek = A * hypotf(rx[0], rx[1]) * hypotf(rx[0], rx[1]) + A * SQ(eE0 / hOmg);

   // ekq = A * (sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ekq = A * hypotf(rx[0] + rx[2], rx[1]) * hypotf(rx[0] + rx[2], rx[1]) + A * SQ(eE0 / hOmg);

   // xk = 2 * A * eE0 * sqrt((rx[0]) ** 2 + (rx[1]) ** 2) / hOmg ** 2
   xk = 2 * A * eE0 * hypotf(rx[0], rx[1]) / SQ(hOmg);

   // xkq = 2 * A * eE0 * sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2) / hOmg ** 2
   xkq = 2 * A * eE0 * hypotf(rx[0] + rx[2], rx[1]) / SQ(hOmg);

   // singmatrix = numba.cuda.shared.array((10,N),dtype=numba.types.complex128)
   int sizesing = N * sizeof(float);
   float* singr, singi;
   checkCudaErrors(hipMalloc((void**)&singr, 6 * sizesing));
   checkCudaErrors(hipMalloc((void**)&singi, 4 * sizesing));

   n = 0
   for (int j=-(N - 1)/2; i < ((N-1)/2+1)); i++) {
      singr[0 + n * 6] = 2 * atan2f(Gamm, ek - hOmg / 2 + hOmg * i);
      singr[1 + n * 6] = 2 * atan2f(Gamm, ekq - hOmg / 2 + hOmg * i);
      singr[2 + n * 6] = 2 * atan2f(Gamm, ek + hOmg / 2 + hOmg * i);
      singr[3 + n * 6] = 2 * atan2f(Gamm, ekq + hOmg / 2 + hOmg * i);

      singr[4 + n * 6] = step(mu - hOmg / 2 - hOmg * i, 0.f); // 8 -> 4
      singr[5 + n * 6] = step(mu + hOmg / 2 - hOmg * i, 0.f); // 9 -> 5

      //------//

      singi[0 + n * 10] = logf(Gammsq + SQ(ek - hOmg / 2 + hOmg * i));  // 4 -> 0
      singi[1 + n * 10] = logf(Gammsq + SQ(ekq - hOmg / 2 + hOmg * i)); // 5 -> 1
      singi[2 + n * 10] = logf(Gammsq + SQ(ek + hOmg / 2 + hOmg * i));  // 6 -> 2
      singi[3 + n * 10] = logf(Gammsq + SQ(ekq + hOmg / 2 + hOmg * i)); // 7 -> 3

      n = n + 1
   }

   int sizedbl = (2*N-1) * sizeof(float);
   float* dblr, dbli;
   checkCudaErrors(hipMalloc((void**)&dblr, 5 * sizedbl));
   checkCudaErrors(hipMalloc((void**)&dbli, 2 * sizedbl));

   hipFloatComplex* dblz;
   checkCudaErrors(hipMalloc((void**)&dblz, 2*(2*N-1)*sizeof(hipFloatComplex)));

   n = 0
   for (int i=-(N - 1); i < N; i++){
       dblr[0 + n * 5] = 2 * atan2f(Gamm, (ek - mu + hOmg * i));
       dblr[1 + n * 5] = 2 * atan2f(Gamm, (ekq - mu + hOmg * i));

       dblr[2 + n * 5] = jnf(i, xk); // Bessel function of order i
       dblr[3 + n * 5] = jnf(i, xkq);

       dblr[4 + n * 5] = ek - ekq + hOmg * i;

       //------//

       dbli[0 + n * 2] = logf(Gammsq + SQ(ek - mu + hOmg * i));
       dbli[1 + n * 2] = logf(Gammsq + SQ(ekq - mu + hOmg * i));

       //------//

       dblz[0 + n * 2] = make_hipFloatComplex(ek - ekq + hOmg * i, 2 * Gamm);
       dblz[1 + n * 2] = make_hipFloatComplex(hOmg * i, 2 * Gamm);
       n = n + 1
    }


   for (int n=0; n<N; n++){
       for (int alpha=0; alpha<N; alpha++){
           for (int beta=0; beta<N; beta++){
               for (int gamma=0; gamma<N; gamma++){
                   for (int s=0; s<N; s++){
                       for (int l=0; l<N; l++){
                           p1p = dblr[4,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblr[0,s + alpha] - singmatrix[4,alpha] + dbli[0,s + alpha])
                           p2p = dblz[0,alpha - gamma + N - 1] * (singmatrix[0,beta] - dblr[0,s + beta] + singmatrix[4,beta] - dbli[0,s + beta])
                           p3p = dblz[1,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblr[1,s + gamma] - singmatrix[5,gamma] + dbli[1,s + gamma])

                           p1m = dblr[4,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblr[0,s + alpha] - singmatrix[6,alpha] + dbli[0,s + alpha])

                           p2m = dblz[0,alpha - gamma + N - 1] * ( singmatrix[2,beta] - dblr[0,s + beta] + singmatrix[6,beta] - dbli[0,s + beta])

                           p3m = dblz[1,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblr[1,s + gamma] - singmatrix[7,gamma] + dbli[1,s + gamma])

                           d1 = -2 * complex(0, 1) * dblr[4,beta - gamma + N - 1] * dblz[0,alpha - gamma + N - 1] * dblz[1,alpha - beta + N - 1]

                           omint1p = singmatrix[8,s] * ((p1p + p2p + p3p) / d1)

                           omint1m = singmatrix[9,s] * ((p1m + p2m + p3m) / d1)

                           bess1 = dblr[3,gamma - n + N - 1] * dblr[3,gamma - l + N - 1] * dblr[2,beta - l + N - 1] * dblr[2,beta - s + N - 1] * dblr[2,alpha - s + N - 1] * dblr[2,alpha - n + N - 1]

                           grgl = bess1 * (omint1p - omint1m)

                           pp1p = dblr[4,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblr[1,s + gamma] - singmatrix[5,gamma] + dbli[1,s + gamma])

                           pp2p = dblz[0,alpha - gamma + N - 1] * (-singmatrix[1,beta] + dblr[1,s + beta] + singmatrix[5,beta] - dbli[1,s + beta])

                           pp3p = dblz[1,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblr[0,s + alpha] - singmatrix[4,alpha] + dbli[0,s + alpha])

                           pp1m = dblr[4,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblr[1,s + gamma] - singmatrix[7,gamma] + dbli[1,s + gamma])

                           pp2m = dblz[0,alpha - gamma + N - 1] * (-singmatrix[3,beta] + dblr[1,s + beta] + singmatrix[7,beta] - dbli[1,s + beta])

                           pp3m = dblz[1,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblr[0,s + alpha] - singmatrix[6,alpha] + dbli[0,s + alpha])

                           d2 = -2 * complex(0, 1) * dblr[4,alpha - beta + N - 1] * dblz[0,alpha - gamma + N - 1] * dblz[1,beta - gamma + N - 1]

                           omint2p = singmatrix[8,s] * ((pp1p + pp2p + pp3p) / d2)

                           omint2m = singmatrix[9,s] * ((pp1m + pp2m + pp3m) / d2)

                           bess2 = dblr[3,gamma - n + N - 1] * dblr[3,gamma - s + N - 1] * dblr[3,beta - s + N - 1] * dblr[3,beta - l + N - 1] * dblr[2,alpha - l + N - 1] * dblr[2,alpha - n + N - 1]

                           glga = bess2 * (omint2p - omint2m)

                           dds = dds + Gamm * (grgl + glga)
                        }
                    }
                }
            }
        }
    }
   return -8 * hipCrealf(dds) / CB(HIP_PI_F);
}
