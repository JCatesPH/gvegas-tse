#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>
#include <assert.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

int main(int argc, char* argv[])
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n "ncall0" -i "itmx0" -a "nacc" -b "nBlockSize0" -d "ndim0"
   //
   // parameters:
   //   ncall = 1024*ncall0 is the amount of function calls
   //   itmx  = itmx0 is the maximum iterations for the algorithm
   //   acc   = nacc*0.00001f is the desired accuracy
   //   nBlockSize = nBlockSize0 is the size of the CUDA block
   //   ndim = ndim0 is the dimension of the integration space

   int ncall0 = 0;
   int itmx0 = 10;
   int nacc  = 1;
   int nBlockSize0 = 256;
   int ndim0 = 3;
   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:d:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
           case 'd':
             ndim0 = atoi(optarg);
             break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = (1 << ncall0)*1024;
   itmx = itmx0;
   acc = (float)nacc*0.000001f;
   nBlockSize = nBlockSize0;
   ndim = ndim0;

   assert(ndim <= ndim_max);

   mds = 1;

   ng = 0;
   npg = 0;

   /*-------- Setting integration limits ---------*/
   /* for (int i=0;i<ndim;i++) {
      xl[i] = 0.;
      xu[i] = 1.;
   }*/

   // Based on original description of problem. 
    //"The integrand is Ds(kx,ky,qx,qy)/(2*pi)^3, and the limits of integration are kx=[-pi/a,pi/a],ky=[-pi/a,pi/a] , qx=[-pi/a,pi/a] and qy=[-pi/a,pi/a]."
    //"For qx and qy it is more efficient to use qx=[0.001,pi/a] and qy=0, because of the symmetry of the problem. kx and ky should be as we said before kx=[-pi/a,pi/a],ky=[-pi/a,pi/a]."

    xl[0] = -3.14159265358979 / 4 // kxi
    xu[0] = 3.14159265358979  / 4 // kxf
 
    xl[1] = -3.14159265358979 / 4 // kyi
    xu[1] = 3.14159265358979f / 4 // kyf
 
    xl[2] = 0.001                // qxi
    xu[2] = 3.14159265358979 / 4 // qxf
 
    /*---------------------------------------------*/
//If nprn = 1 it prints the whole work, when nprn = 0, just the text in this code.
//If nprn = -1, we can get the grid update information.

  nprn = 1;
//  nprn = -1;
//  nprn = 0;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   myVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   //Print running times!
   std::cout<<"#==========================="<<std::endl;
   printf("# Function call time per iteration: %lf\n", timeVegasCallAndFill/(double)it);
   printf("# Refining time per iteration: %lf\n", timeVegasRefine/(double)it);
   std::cout<<"#==========================="<<std::endl;


    /* Instructions for performance measure
    char archivo[64];
    sprintf(archivo, "./datos/testtoolbox/prodexp.dat");
    FILE *f = fopen(archivo, "ab+");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    fprintf(f, "%d %d %.9lf %.9lf %lf\n", ndim0, ncall0, avgi, sd, timeVegasCallAndFill+timeVegasRefine);
    fclose(f);
    */
   return 0;
}
