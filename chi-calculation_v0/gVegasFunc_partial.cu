#include "hip/hip_runtime.h"
#include "vegasconst.h"

#include <hip/hip_complex.h> // Complex number module of cuda.

#define HIP_PI_F 3.141592654f

/*-------- constants for chi ---------*/
#define mu      0.1f
#define hOmg    0.3f
#define a       4.f
#define A       4.f
#define rati    0.3
#define eE0     rati * (hOmg * hOmg) / (2 * sqrt(A * mu))
#define Gamm    0.003
#define KT      1e-6
#define shift   A * (eE0 / hOmg) * (eE0 / hOmg)
#define Gammsq  Gamm * Gamm
#define N       3

/*-------- helpful macros ---------*/
#define SQ(x)  (x * x) // Squares the argument
#define CB(x)  (x * x * x) // Cubes the argument

__device__
float chi(float* rx, float wgt)
{
   float dds = 0.f;
   // ds = 0  // UNUSED
   float ek;
   float ekq;
   float xk;
   float xkq;

   // ek = A * (sqrt((rx[0]) ** 2 + (rx[1]) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ek = A * hypotf(rx[0], rx[1]) * hypotf(rx[0], rx[1]) + A * SQ(eE0 / hOmg);

   // ekq = A * (sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2)) ** 2 + A * (eE0 / hOmg) ** 2
   ekq = A * hypotf(rx[0] + rx[2], rx[1]) * hypotf(rx[0] + rx[2], rx[1]) + A * SQ(eE0 / hOmg);

   // xk = 2 * A * eE0 * sqrt((rx[0]) ** 2 + (rx[1]) ** 2) / hOmg ** 2
   xk = 2 * A * eE0 * hypotf(rx[0], rx[1]) / SQ(hOmg);

   // xkq = 2 * A * eE0 * sqrt((rx[0] + qx) ** 2 + (rx[1] + 0) ** 2) / hOmg ** 2
   xkq = 2 * A * eE0 * hypotf(rx[0] + rx[2], rx[1]) / SQ(hOmg);

   // singmatrix = numba.cuda.shared.array((10,N),dtype=numba.types.complex128)
   int sizesing = N * sizeof(float);
   float* singreal, singimag;
   checkCudaErrors(hipMalloc((void**)&singreal, 6 * sizesing));
   checkCudaErrors(hipMalloc((void**)&singrimag, 4 * sizesing));

   n = 0
   for (int j=-(N - 1)/2; i < ((N-1)/2+1)); i++) {
      singreal[0 + n * 6] = 2 * atan2f(Gamm, ek - hOmg / 2 + hOmg * i);
      singreal[1 + n * 6] = 2 * atan2f(Gamm, ekq - hOmg / 2 + hOmg * i);
      singreal[2 + n * 6] = 2 * atan2f(Gamm, ek + hOmg / 2 + hOmg * i);
      singreal[3 + n * 6] = 2 * atan2f(Gamm, ekq + hOmg / 2 + hOmg * i);

      singreal[4 + n * 6] = step(mu - hOmg / 2 - hOmg * i, 0.f); // 8 -> 4
      singreal[5 + n * 6] = step(mu + hOmg / 2 - hOmg * i, 0.f); // 9 -> 5

      //------//

      singimag[0 + n * 10] = logf(Gammsq + SQ(ek - hOmg / 2 + hOmg * i));  // 4 -> 0
      singimag[1 + n * 10] = logf(Gammsq + SQ(ekq - hOmg / 2 + hOmg * i)); // 5 -> 1
      singimag[2 + n * 10] = logf(Gammsq + SQ(ek + hOmg / 2 + hOmg * i));  // 6 -> 2
      singimag[3 + n * 10] = logf(Gammsq + SQ(ekq + hOmg / 2 + hOmg * i)); // 7 -> 3

      n = n + 1
   }

   int sizedbl = (2*N-1) * sizeof(float);
   float* dblreal, dblimag;
   checkCudaErrors(hipMalloc((void**)&dblreal, 5 * sizedbl));
   checkCudaErrors(hipMalloc((void**)&dblimag, 2 * sizedbl));

   hipFloatComplex* dblcomplex;
   checkCudaErrors(hipMalloc((void**)&dblcomplex, 2*(2*N-1)*sizeof(hipFloatComplex)));

   n = 0
   for (int i=-(N - 1); i < N; i++){
       dblreal[0 + n * 5] = 2 * atan2f(Gamm, (ek - mu + hOmg * i));
       dblreal[1 + n * 5] = 2 * atan2f(Gamm, (ekq - mu + hOmg * i));

       dblreal[2 + n * 5] = jnf(i, xk); // Bessel function of order i
       dblreal[3 + n * 5] = jnf(i, xkq);

       dblreal[4 + n * 5] = ek - ekq + hOmg * i;

       //------//

       dblimag[0 + n * 2] = logf(Gammsq + SQ(ek - mu + hOmg * i));
       dblimag[1 + n * 2] = logf(Gammsq + SQ(ekq - mu + hOmg * i));

       //------//

       dblcomplex[0 + n * 2] = make_hipFloatComplex(ek - ekq + hOmg * i, 2 * Gamm);
       dblcomplex[1 + n * 2] = make_hipFloatComplex(hOmg * i, 2 * Gamm);
       n = n + 1
    }


   for (int n=0; n<N; n++){
       for (int alpha=0; alpha<N; alpha++){
           for (int beta=0; beta<N; beta++){
               for (int gamma=0; gamma<N; gamma++){
                   for (int s=0; s<N; s++){
                       for (int l=0; l<N; l++){
                           p1p = dblmatrix[6,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblmatrix[0,s + alpha] - singmatrix[4,alpha] + dblmatrix[2,s + alpha])
                           p2p = dblmatrix[7,alpha - gamma + N - 1] * (singmatrix[0,beta] - dblmatrix[0,s + beta] + singmatrix[4,beta] - dblmatrix[2,s + beta])
                           p3p = dblmatrix[8,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblmatrix[1,s + gamma] - singmatrix[5,gamma] + dblmatrix[3,s + gamma])

                           p1m = dblmatrix[6,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblmatrix[0,s + alpha] - singmatrix[6,alpha] + dblmatrix[2,s + alpha])

                           p2m = dblmatrix[7,alpha - gamma + N - 1] * ( singmatrix[2,beta] - dblmatrix[0,s + beta] + singmatrix[6,beta] - dblmatrix[2,s + beta])

                           p3m = dblmatrix[8,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblmatrix[1,s + gamma] - singmatrix[7,gamma] + dblmatrix[3,s + gamma])

                           d1 = -2 * complex(0, 1) * dblmatrix[6,beta - gamma + N - 1] * dblmatrix[7,alpha - gamma + N - 1] * dblmatrix[8,alpha - beta + N - 1]

                           omint1p = singmatrix[8,s] * ((p1p + p2p + p3p) / d1)

                           omint1m = singmatrix[9,s] * ((p1m + p2m + p3m) / d1)

                           bess1 = dblmatrix[5,gamma - n + N - 1] * dblmatrix[5,gamma - l + N - 1] * dblmatrix[4,beta - l + N - 1] * dblmatrix[4,beta - s + N - 1] * dblmatrix[4,alpha - s + N - 1] * dblmatrix[4,alpha - n + N - 1]

                           grgl = bess1 * (omint1p - omint1m)

                           pp1p = dblmatrix[6,alpha - beta + N - 1] * (-singmatrix[1,gamma] + dblmatrix[1,s + gamma] - singmatrix[5,gamma] + dblmatrix[3,s + gamma])

                           pp2p = dblmatrix[7,alpha - gamma + N - 1] * (-singmatrix[1,beta] + dblmatrix[1,s + beta] + singmatrix[5,beta] - dblmatrix[3,s + beta])

                           pp3p = dblmatrix[8,beta - gamma + N - 1] * (singmatrix[0,alpha] - dblmatrix[0,s + alpha] - singmatrix[4,alpha] + dblmatrix[2,s + alpha])

                           pp1m = dblmatrix[6,alpha - beta + N - 1] * (-singmatrix[3,gamma] + dblmatrix[1,s + gamma] - singmatrix[7,gamma] + dblmatrix[3,s + gamma])

                           pp2m = dblmatrix[7,alpha - gamma + N - 1] * (-singmatrix[3,beta] + dblmatrix[1,s + beta] + singmatrix[7,beta] - dblmatrix[3,s + beta])

                           pp3m = dblmatrix[8,beta - gamma + N - 1] * (singmatrix[2,alpha] - dblmatrix[0,s + alpha] - singmatrix[6,alpha] + dblmatrix[2,s + alpha])

                           d2 = -2 * complex(0, 1) * dblmatrix[6,alpha - beta + N - 1] * dblmatrix[7,alpha - gamma + N - 1] * dblmatrix[8,beta - gamma + N - 1]

                           omint2p = singmatrix[8,s] * ((pp1p + pp2p + pp3p) / d2)

                           omint2m = singmatrix[9,s] * ((pp1m + pp2m + pp3m) / d2)

                           bess2 = dblmatrix[5,gamma - n + N - 1] * dblmatrix[5,gamma - s + N - 1] * dblmatrix[5,beta - s + N - 1] * dblmatrix[5,beta - l + N - 1] * dblmatrix[4,alpha - l + N - 1] * dblmatrix[4,alpha - n + N - 1]

                           glga = bess2 * (omint2p - omint2m)

                           dds = dds + Gamm * (grgl + glga)
                        }
                    }
                }
            }
        }
    }
   return -8 * hipCrealf(dds) / CB(HIP_PI_F);
}
