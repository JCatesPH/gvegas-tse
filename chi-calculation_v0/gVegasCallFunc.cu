#include "hip/hip_runtime.h"
#include "vegasconst.h"
#include "vegas.h"

__global__
void gVegasCallFunc(float* gFval, int* gIAval)
{
   //--------------------
   // Check the thread ID
   //--------------------
   const unsigned int tIdx  = threadIdx.x;
   const unsigned int bDimx = blockDim.x;

   const unsigned int bIdx  = blockIdx.x;
   const unsigned int gDimx = gridDim.x;
   const unsigned int bIdy  = blockIdx.y;
   //   const unsigned int gDimy = gridDim.y;

   unsigned int bid  = bIdy*gDimx+bIdx;
   const unsigned int tid = bid*bDimx+tIdx;

   //   int ipg = tid%g_npg;
   int ig = tid/g_npg;

   unsigned nCubeNpg = g_nCubes*g_npg;

   if (tid<nCubeNpg) {

      unsigned ia[ndim_max];
      
      unsigned int tidRndm = tid;
      
      int kg[ndim_max];
      
      unsigned igg = ig;
      for (int j=0;j<g_ndim;j++) {
         kg[j] = igg%g_ng+1;
         igg /= g_ng;
      }
      
      //            randa(g_ndim,randm);
      float randm[ndim_max];
      fxorshift128(tidRndm, g_ndim, randm);
      
      float x[ndim_max];
      
      float wgt = g_xjac;
      for (int j=0;j<g_ndim;j++) {
         float xo,xn,rc;
         xn = (kg[j]-randm[j])*g_dxg+1.f;
         ia[j] = (int)xn-1;
         if (ia[j]<=0) {
            xo = g_xi[j][ia[j]];
            rc = (xn-(float)(ia[j]+1))*xo;
         } else {
            xo = g_xi[j][ia[j]]-g_xi[j][ia[j]-1];
            rc = g_xi[j][ia[j]-1]+(xn-(float)(ia[j]+1))*xo;
         }
         x[j] = g_xl[j]+rc*g_dx[j];
         wgt *= xo*(float)g_nd;
      }
      
      float f;   

	   f = wgt * chi(x, g_ndim);
      //      gFval[tid] = (float)typeFinal[2];
      gFval[tid] = f;
      for (int idim=0;idim<g_ndim;idim++) {
         gIAval[idim*nCubeNpg+tid] = ia[idim];
      }
   }

}
