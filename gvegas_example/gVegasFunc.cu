#include "hip/hip_runtime.h"
#include "vegasconst.h"

__device__
double func(double* rx, double wgt)
{

   double value = 0;
   double result = 0;
   // double T = 20; // For the F-D Distribution
   // double mu = 1;
   double c = 0.01; // Constant for M-B Dis. that equals ratio : m / k_B T

   for (int i=0;i<g_ndim;i++) {
      value += rx[i] * rx[i];
   }
   
   // Just return the sum the vector.
   // result = value;

   // sin of vector's sum
   // result = sin(value);

   // Maxwell-Boltzmann Distribution
   result = sqrt(2 * c * c * c / 3.14159) * value * exp(-c / 2 * value);

   // Fermi-Dirac Distribution
   // result = 1 / ( exp((value - mu) / (T)) + 1 );

   return result;

}

/*
__device__
double func(double* rx, double wgt)
{
   double value = 1.;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.*rx[i];
   }
   return value;

}
*/

