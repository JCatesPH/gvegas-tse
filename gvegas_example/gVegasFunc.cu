#include "hip/hip_runtime.h"
#include "vegasconst.h"
#include "const.h"

__device__
double func(double* rx, double wgt)
{

   double value = 0;
   double result = 0;
   // double c = 0.01; // Constant for M-B Dis. that equals ratio : m / k_B T
   // double T = 20; // For the F-D Distribution
   // double mu = 1;
   // double sig = 1.0; // For the Gaussian Distribution
   // double mu  = 0.0;

   for (int i=0;i<g_ndim;i++) {
      // value += rx[i]; // Simple sum (for F-D)
      // value *= rx[i]; // Simple product
      // value += rx[i] * rx[i]; // Sum of squares (for M-B or Singular)
      // value += (rx[i] - mu) * (rx[i] - mu);
      value += cos(log(rx[i]) / rx[i]) / rx[i]
   }
   
   // Just return the sum the vector.
   result = value;

   // sin of vector's sum
   // result = sin(value);

   // Maxwell-Boltzmann Distribution
   // result = sqrt(2 * c * c * c / PI) * value * exp(-c / 2 * value);

   // Fermi-Dirac Distribution
   // result = 1 / ( exp((value - mu) / (T)) + 1 );

   // Gaussian Distribution
   // result = 1 / (sqrt(2 * PI * sig * sig)) * exp(-value / (2 * sig * sig));

   // Singular Example from Mathematica (Numerator determines spreading)
   // result = 1 / sqrt(value);

   return result;

}

/*
__device__
double func(double* rx, double wgt)
{
   double value = 1.;
   for (int i=0;i<g_ndim;i++) {
      value *= 2.*rx[i];
   }
   return value;

}
*/

