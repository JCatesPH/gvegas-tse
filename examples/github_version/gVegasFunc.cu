#include "hip/hip_runtime.h"
#include "vegasconst.h"
#define HIP_PI_F 3.141592654f

__device__
float func(float* rx, float wgt)
{

   float value = 0.f;
   for (int i=0;i<g_ndim;i++) {
      value += rx[i]*rx[i];
   }
   return value;
}

__device__
float oscillate(float* rx, float wgt, float* dillate, float* offset)
{
   float value = 0.f;
   for (int i = 0; i < g_ndim; i++) {
      value += dillate[i] * rx[i];
   }
   value += HIP_PI_F * offset[0];
   value = cosf(value);
   return value;
}

__device__
float prodpeak(float* rx, float wgt, float* move, float* offset)
{
   float value = 1.f;
   for (int i = 0; i < g_ndim; i++) {
      value *= 1.f / ((rx[i]-move[i])*(rx[i]-move[i]) + (1.f/offset[i])*(1.f/offset[i]));
   }
   return value;
}

__device__
float cornerpeak(float* rx, float wgt, float* offset)
{
   float value = 1.f;
   for (int i = 0; i < g_ndim; i++) {
      value += offset[i] * rx[i];
   }
   value = 1.f / powf(value, (float)(g_ndim+1));
   return value;
}

__device__
float gaussian(float* rx, float wgt, float* move, float* offset)
{
   float value = 0.f;
   for (int i = 0; i < g_ndim; i++) {
      value = value - offset[i]*offset[i]*(rx[i]-move[i])*(rx[i]-move[i]);
   }
   value = expf(value);
   return value;
}

__device__
float czerocont(float* rx, float wgt, float* move, float* offset)
{
   float value = 0.f;
   for (int i = 0; i < g_ndim; i++) {
      value = value - offset[i]*fabsf(rx[i]-move[i]);
   }
   value = expf(value);
   return value;
}

__device__
float discont(float* rx, float wgt, float* limit, float* offset)
{
   float value = 0.f;
   for (int i = 0; i < g_ndim; i++) {
      value += offset[i]*rx[i];
   }
   value = expf(value) * (float)((1-(rx[0] > limit[0]))*(1-(rx[1] > limit[0])));
   return value;
}
