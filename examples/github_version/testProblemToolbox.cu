#include "hip/hip_runtime.h"
#include "vegasconst.h"
#define HIP_PI_F 3.141592654f

__device__
float sum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i];
	}
	value = 1.f / sqrt((float)dim/12.f) * (value - (float)dim / 2.f);
	return value;
}

__device__
float sqsum(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += rx[i] * rx[i];
	}
	value = sqrtf(45.f / (4.f * (float)dim)) * (value - (float)dim / 3);
	return value;
}

__device__
float sumsqroot(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		value += sqrtf(rx[i]);
	}
	value = sqrtf(18.f / (float)dim) * (value - 2.f/3.f * (float)dim);
	return value;
}

__device__
float prodones(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= copysignf(1.f, rx[i]-0.5f);
	}
	return value;
}

__device__
float prodexp(float* rx, int dim)
{
	float e = sqrtf((15.f * expf(15.f) + 15.f) / (13.f * expf(15.f) + 17.f));
	e = powf(e, float(dim) * 0.5f);
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= ((expf(30.f * rx[i] - 15.f)) - 1.f) / (expf(30.f * rx[i] - 15.f) + 1.f);		
	}
	value *= e;
	return value;
}

__device__
float prodcub(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= (-2.4f*sqrtf(7.f)*(rx[i]-0.5f)+8.f*sqrtf(7.f)*(rx[i]-0.5f)*(rx[i]-0.5f)*(rx[i]-0.5f));
	}
	return value;
}

__device__
//PRODX has a lot of extremes when dimensions are big, it's expected to not do well
float prodx(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= (rx[i] - 0.5f);
	}
	value *= powf(2.f*sqrtf(3.f), (float) dim);
	return value;
}

__device__
float sumfifj(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 0; i < dim; i++){
		float aux = 0.f;
		for (int j = 0; j < i; j++){
			aux += copysignf(1.f,(1.f/6.f-rx[j])*(rx[j]-4.f/6.f));
		}
		value += copysignf(1.f,(1.f/6.f-rx[i])*(rx[i]-4.f/6.f))*aux;
	}
	value *= sqrtf(2.f/(float)(dim*(dim-1)));
	return value;
}

__device__
float sumfonefj(float* rx, int dim)
{
	float value = 0.f;
	for (int i = 1; i < dim; i++){
		value += 27.20917094*rx[i]*rx[i]*rx[i]-36.1925085*rx[i]*rx[i]+8.983337562*rx[i]+0.7702079855;
	}
	value *= (27.20917094*rx[0]*rx[0]*rx[0]-36.1925085*rx[0]*rx[0]+8.983337562*rx[0]+0.7702079855)/sqrtf((float)dim-1.f);
	return value;
}

__device__
float hellekalek(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= ((rx[i] - 0.5f)/sqrtf(12.f));
	}
	return value;
}

__device__
float roosarnoldone(float* rx, int dim)
{
	float value = 1.f/(float)dim;
	float aux = 0.f;
	for (int i = 0; i < dim; i++){
		aux += fabsf(4.f*rx[i]-2.f)-1.f;
	}
	value *= aux;
	return value;
}

__device__
//Can give huge error
float roosarnoldtwo(float* rx, int dim)
{
	float value = sqrtf(1.f/(powf(4.f/3.f, (float)dim)-1.f));
	for (int i = 0; i < dim; i++){
		value *= (fabsf(4.f*rx[i]-2.f) - 1.f);
	}
	return value;
}

__device__
float roosarnoldthree(float* rx, int dim)
{
	float value = 1.f/sqrtf(powf(HIP_PI_F*HIP_PI_F/8.f, (float)dim)-1.f);
	for (int i = 0; i < dim; i++){
		value *= (HIP_PI_F/2.f*sinf(HIP_PI_F*rx[i])-1.f);
	}
	return value;
}

__device__
//Choosing only RST1, since it's the most difficult.
float rst(float* rx, int dim)
{
	float value = 1.f/sqrtf(powf(1.f+1.f/12.f,(float)dim)-1.f);
	for (int i = 0; i < dim; i++){
		value *= ((fabsf(4.f*rx[i]-2.f)+1.f)/2.f-1.f);
	}
	return value;
}

__device__
float sobolprod(float* rx, int dim)
{
	float value = 1.f;
	for (int i = 0; i < dim; i++){
		value *= (1.f+1.f/((float)(3*(i+2)*(i+2))));
	}
	value = sqrtf(1.f/(value-1.f));
	for (int i = 0; i < dim; i++){
		value *= ((float)(i+1)+2.f*rx[i])/(float)(i + 2)-1.f;
	}
	return value;
}

__device__
//Choosing beta = 1 and alpha_i = 1 for every i.
float oscill(float* rx, int dim)
{
	float value = 2.f*HIP_PI_F;
	float p = 1.f;
	for (int i = 0; i < dim; i++){
		value += rx[i];
		p *= sinf(0.5f);
	}
	value = cosf(value)-powf(2.f, (float)dim)*cosf(2.f*HIP_PI_F+0.5f*(float)dim)*p;
	return value;
}

__device__
//Choosing beta_i = 0.5 and alpha_i = 1 for every i.
float prpeak(float* rx, int dim)
{
	float value = 1.f;
	float e = 1.f;
	for (int i = 0; i < dim; i++){
		value *= 1.f/(1+(rx[i]-0.5f)*(rx[i]-0.5f));
		e *= (atanf(0.5f)-atanf(-0.5f));
	}
	value += -e;
	return value;
}

//There are 4 functions missing from the document (CORPEAK, GAUSSIAN, C0 and DISCONT), but it gets really hard from here on to estimate numbers and I prefer stopping here. 17 is a good enough number of test functions.
