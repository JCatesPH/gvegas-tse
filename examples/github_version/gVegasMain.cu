#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>

// includes, project
#include "hip/hip_runtime_api.h"
// include initial files

#define __MAIN_LOGIC
#include "vegas.h"
#include "gvegas.h"
#undef __MAIN_LOGIC

#include "kernels.h"

int main(int argc, char** argv)
{

   //------------------
   //  Initialization
   //------------------
   //
   // program interface:
   //   program -n="ncall0" -i="itmx0" -a="nacc" -b="nBlockSize0"
   //
   // parameters:
   //   ncall = 1024*ncall0
   //   itmx  = itmx0
   //   acc   = nacc*0.00001f
   //   nBlockSize = nBlockSize0
   //

   int ncall0 = 0;
   int itmx0 = 10;
   int nacc  = 1;
   int nBlockSize0 = 256;
   int ndim0 = 6;
   int c;

   while ((c = getopt (argc, argv, "n:i:a:b:d:")) != -1)
       switch (c)
         {
         case 'n':
           ncall0 = atoi(optarg);
           break;
         case 'i':
           itmx0 = atoi(optarg);
           break;
         case 'a':
           nacc = atoi(optarg);
           break;
         case 'b':
           nBlockSize0 = atoi(optarg);
           break;
         case 'd':
           ndim0 = atoi(optarg);
           break;
         case '?':
           if (isprint (optopt))
             fprintf (stderr, "Unknown option `-%c'.\n", optopt);
           else
             fprintf (stderr,
                      "Unknown option character `\\x%x'.\n",
                      optopt);
           return 1;
         default:
           abort ();
         }

   ncall = (1 << ncall0)*1024;
   itmx = itmx0;
   acc = (float)nacc*0.000001f;
   nBlockSize = nBlockSize0;
   ndim = ndim0;

   assert(ndim <= ndim_max);

   mds = 1;

   ng = 0;
   npg = 0;

   for (int i=0;i<ndim;i++) {
      xl[i] = 0.;
      xu[i] = 1.;
   }
   //If nprn = 1 it prints the whole work, when nprn = 0, just the text in this code
   //If nprn = -1, we can get the grid update information.

   nprn = 1;
//   nprn = -1;
//  nprn = 0;

   double avgi = 0.;
   double sd = 0.;
   double chi2a = 0.;

   gVegas(avgi, sd, chi2a);

   //-------------------------
   //  Print out information
   //-------------------------
   std::cout.clear();
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of Thread Block Size : "<<nBlockSize<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
   std::cout<<"# No. of func calls / iter : "<<ncall<<std::endl;
   std::cout<<"# No. of max. iterations   : "<<itmx<<std::endl;
   std::cout<<"# Desired accuracy         : "<<acc<<std::endl;
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
   std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   hipDeviceReset();

   //Print running times!
   std::cout<<"#==========================="<<std::endl;
   std::cout<<"# Function call time per iteration: " <<timeVegasCall/(double)it<<std::endl;
   std::cout<<"# Values moving time per iteration: " <<timeVegasMove/(double)it<<std::endl;
   std::cout<<"# Filling (reduce) time per iteration: " <<timeVegasFill/(double)it<<std::endl;
   std::cout<<"# Refining time per iteration: " <<timeVegasRefine/(double)it<<std::endl;
   std::cout<<"#==========================="<<std::endl;

   /* Instructions for time measure
   
    int qth;
    qth = omp_get_max_threads();
    printf("%d \n", qth);
    char archivo[64];
    sprintf(archivo, "./datos/redtime/gVegas%d/red_d%dn%d.dat", qth, ndim0, ncall0);
    FILE *f = fopen(archivo, "ab+");
    if (f == NULL)
    {
        printf("Error opening file!\n");
        exit(1);
    }

    fprintf(f, "%lf\n", (timeVegasCall+timeVegasMove+timeVegasFill)/(double)it);
    fclose(f);
    */

   return 0;
}
