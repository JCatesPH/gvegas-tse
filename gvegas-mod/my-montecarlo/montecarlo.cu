#include <iostream>
#include <iomanip>
#include <cmath>
#include <omp.h>
#include <assert.h>

#include "hip/hip_runtime_api.h"

#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

void mckernel(double& avgi, double& sd)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(g_ndim), &ndim, sizeof(int)));
    return;
}