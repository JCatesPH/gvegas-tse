#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <unistd.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>
#include <assert.h>

#include "hip/hip_runtime_api.h"

// This file contains the interface to the Monte Carlo program.


void main()
{
    /*
    The following parameters are set by the user:
        - tpb : CUDA Threads per Block.
        - maxit : Maximum number of times that the integration will be computed.
        - cpi : Number of function calls per iteration.
        - ndim : Number of dimensions in your integration.
    */ 

    int tpb = 512;
    int maxit = 20;
    int ndim = 5;
    int cpi = 1024 * 32

    assert(ndim <= ndim_max);

    // The limits of integration are set here. xl is lower bound. xu is upper bound.
    for (int i=0;i<ndim;i++) { 
        xl[i] = 1.; 
        xu[i] = 10.; 
    }

    // The result and standard deviation are instantiated.
    double avgi = 0.;
    double sd = 0.;
 
    // Integration function is called.
    time_t tic;

    MonteCarlo(avgi, sd);
 
    time_t toc;

    double seconds = difftime(toc, tic);

    //-------------------------
    //  Print out information
    //-------------------------
    std::cout.clear();
    std::cout<<"#==========================="<<std::endl;
    std::cout<<"# No. of Threads per Block : "<<tpd<<std::endl;
    std::cout<<"#==========================="<<std::endl;
    std::cout<<"# No. of dimensions        : "<<ndim<<std::endl;
    std::cout<<"# No. of func calls / iter : "<<cpi<<std::endl;
    std::cout<<"# No. of max. iterations   : "<<maxit<<std::endl;
    std::cout<<"#==========================="<<std::endl;
    std::cout<<"# Answer                   : "<<avgi<<" +- "<<sd<<std::endl;
    //std::cout<<"# Chisquare                : "<<chi2a<<std::endl;
    std::cout<<"#==========================="<<std::endl;
 
    //Print running times!
    std::cout<<"#==========================="<<std::endl;
    //printf("# Function call time per iteration: %lf\n", timeVegasCallAndFill/(double)it);
    //printf("# Refining time per iteration: %lf\n", timeVegasRefine/(double)it);
    std::cout<<"Time to integrate          :"<<seconds<<std::endl;
    std::cout<<"#==========================="<<std::endl;

}