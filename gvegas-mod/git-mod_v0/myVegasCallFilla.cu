#include "hip/hip_runtime.h"
#include "vegasconst.h"
#include "vegas.h"

__device__ float d[ndim_max][nd_max];
__device__ float dti, dtsi;
__device__ double doubleti, doubletsi;


__global__
void initzero(void){

/*
  for (int dim = 0; dim < g_ndim; dim++){
    for (int box = 0; box < g_nd; box++){
      d[dim][box] = 0.0f;
    }
  }
*/

  // Dos alternativas, cudamemset o armar un 0 para cada hilo y llamar bien al kernel 
  d[threadIdx.x][threadIdx.y] = 0.0f;
  

  dti = 0.0f;
  dtsi = 0.0f;
}


__global__
void myVegasCallFilla(int mds)
{
   //--------------------
   // Check the thread ID
   //--------------------
   const unsigned int tIdx  = threadIdx.x;
   const unsigned int bDimx = blockDim.x;

   const unsigned int bIdx  = blockIdx.x;
   const unsigned int gDimx = gridDim.x;
   const unsigned int bIdy  = blockIdx.y;
   //   const unsigned int gDimy = gridDim.y;

   unsigned int bid  = bIdy*gDimx+bIdx;
   const unsigned int tid = bid*bDimx+tIdx;

   //Using float for now, atomicAdd doesn't support double yet...
   __shared__ float block_fb;
   __shared__ float block_f2b;
   __shared__ float block_d[ndim_max][nd_max];


   block_fb = 0.0f;
   block_f2b = 0.0f;
/*   
   for (int idim = 0; idim < g_ndim; idim++){
     for (int ind = 0; ind < g_nd; ind ++)
     block_d[idim][ind] = 0.0f;
   }
*/
   /* Alternative for above  */
      for (int i = 0; i < (g_ndim * g_nd - 1) / bDimx + 1; i++){
        int xdim = (i * bDimx + tIdx) / g_nd;
	int xind = (i * bDimx + tIdx) % g_nd;
	if (xdim < g_ndim){
	  block_d[xdim][xind] = 0.0f;
	}
      }



   //int ig = tid;
   int lane = tIdx % warpSize;
   //d[tid] = 0.0;
   int kg[ndim_max];
   unsigned ia[ndim_max];
   //fb and f2b will be the accumulations of f and the "error", these values
   //will be reduced later and stored in dti and dtsi.
   float f, f2;
   float fb = 0.0f;
   float f2b = 0.0f;


   if (tid<g_nCubes) {

      for (int point = 0; point < g_npg; point++){
        unsigned int tidRndm = tid * g_npg + point;

        unsigned igg = tid;
        for (int j=0;j<g_ndim;j++) {
           kg[j] = igg%g_ng+1;
           igg /= g_ng;
        }

        //Generate a random point in [0,1]^ndim.
        float randm[ndim_max];
        fxorshift128(tidRndm, g_ndim, randm);

        float x[ndim_max];

        float wgt = g_xjac;

        /*
        This piece of code places the random point in the domain of integration,
        g_xi will change at every iteration as a result of the refining step, so
        the weight will change as well.
        */

        for (int j=0;j<g_ndim;j++) {
          float xo,xn,rc;
          xn = (kg[j]-randm[j])*g_dxg+1.f;
          ia[j] = (int)xn-1;
          if (ia[j]<=0) {
            xo = g_xi[j][ia[j]];
            rc = (xn-(float)(ia[j]+1))*xo;
          } else {
            xo = g_xi[j][ia[j]]-g_xi[j][ia[j]-1];
            rc = g_xi[j][ia[j]-1]+(xn-(float)(ia[j]+1))*xo;
          }
          x[j] = g_xl[j]+rc*g_dx[j];
          wgt *= xo*(float)g_nd;
        }


/* Different calls for different functions */	
	f = wgt * func(x, g_ndim);	

	fb += f;
        f2 = f*f;
        f2b += f2;

        //If mds = 1, we just have to add f^2 to the corresponding space in d.
        if (mds > 0){
          for (int idim = 0; idim < g_ndim; idim++) {
            atomicAdd(&block_d[idim][ia[idim]], f2);
          }
        }
      }

      /*When mds = -1, original code uses the data of the first element of the
      cube to store f2b in d, that won't change much if I use the last element.
      If it does, maybe we can go for a decreasing loop in npg...*/
      f2b = sqrt(f2b * g_npg);
      f2b = (f2b - fb) * (f2b - fb);
      if (mds < 0){
        for (int idim = 0; idim < g_ndim; idim++){
          atomicAdd(&block_d[idim][ia[idim]], f2b);
        }
      }
      __syncthreads();

      //REDUCE TIME!!!
      #pragma unroll
      for (int offset = warpSize/2; offset > 0; offset /= 2){
        fb += __shfl_down(fb, offset);
        f2b += __shfl_down(f2b, offset);
      }

      if (0 == lane){
        atomicAdd(&block_fb, fb);
        atomicAdd(&block_f2b, f2b);
      }

      __syncthreads();

      if (0 == tIdx){
        atomicAdd(&dti, block_fb);
        doubleti = (double)dti;
      }

      if (32 == tIdx){
	atomicAdd(&dtsi, block_f2b);
        doubletsi = (double)dtsi;
      }

/* Threaded binning, much better performance */
      for (int i = 0; i < (g_ndim * g_nd - 1) / bDimx + 1; i++){
        int xdim = (i * bDimx + tIdx) / g_nd;
	int xind = (i * bDimx + tIdx) % g_nd;
	if (xdim < g_ndim){
	  atomicAdd(&d[xdim][xind], block_d[xdim][xind]);
	}
      }

/* Sequential binning, low performance :(       
      for (int idim = 0; idim < g_ndim; idim++){
        for (int ind = 0; ind < g_nd; ind ++){
          atomicAdd(&d[idim][ind], block_d[idim][ind]);
	}
      }
*/     
    }

}
